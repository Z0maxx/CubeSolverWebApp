#include "hip/hip_runtime.h"
#include "pll_orient_sequence_executor.cuh"

__device__ void executePLLOrientSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const int idx, const int roundIdx)
{
	for (int i = 0; i < 4; i++)
	{
		dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][roundIdx][i] = const_PLLOrientSequence[i];
		const TranslatedNotation move = translateNotation(const_PLLOrientSequence[i]);
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, move.cubeLayer, move.direction, move.twice);
	}
}