#include "hip/hip_runtime.h"
#include "pll_cycle_sequence_executor.cuh"

__device__ void executePLLCycleSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Notation sequence[], const int idx)
{
	for (int i = 0; i < 8; i++)
	{
		dev_PLLCycleLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][i] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, move.cubeLayer, move.direction, move.twice);
	}
}