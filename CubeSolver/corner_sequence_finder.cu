#include "hip/hip_runtime.h"
#include "corner_sequence_finder.cuh"

__device__ const Notation* findCornerSequence(CornerSolveSequence sequences[24], const uint2 edge, const Color colors[6], const Color targetColor)
{
	for (int i = 0; i < 24; i++)
	{
		if (sequences[i].layer == edge.x && sequences[i].cube == edge.y && colors[sequences[i].side] == targetColor)
		{
			return sequences[i].sequence;
		}
	}
	return { 0 };
}