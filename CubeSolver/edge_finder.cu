#include "hip/hip_runtime.h"
#include "edge_finder.cuh"

__constant__ const Edge const_edges[12] =
{
	{
		Layer_Front,
		Cube_Top,
		{ Side_Top, Side_Front }
	},
	{
		Layer_Front,
		Cube_Left,
		{ Side_Left, Side_Front }
	},
	{
		Layer_Front,
		Cube_Right,
		{ Side_Right, Side_Front }
	},
	{
		Layer_Middle,
		Cube_Top_Left,
		{ Side_Left, Side_Top }
	},
	{
		Layer_Middle,
		Cube_Top_Right,
		{ Side_Right, Side_Top }
	},
	{
		Layer_Back,
		Cube_Top,
		{ Side_Top, Side_Back }
	},
	{
		Layer_Back,
		Cube_Left,
		{ Side_Left, Side_Back }
	},
	{
		Layer_Back,
		Cube_Right,
		{ Side_Right, Side_Back}
	},
	{
		Layer_Front,
		Cube_Bottom,
		{ Side_Bottom, Side_Front}
	},
	{
		Layer_Middle,
		Cube_Bottom_Left,
		{ Side_Left, Side_Bottom}
	},
	{
		Layer_Middle,
		Cube_Bottom_Right,
		{ Side_Right, Side_Bottom}
	},
	{
		Layer_Back,
		Cube_Bottom,
		{ Side_Bottom, Side_Back}
	},
};

__device__ const uint2 findEdge(const Color cubeColors[3][9][6], const Color colors[2], const int count)
{
	for (int i = 0; i < count; i++)
	{
		const Layer layer = const_edges[i].layer;
		const Cube cube = const_edges[i].cube;
		Color targetColors[2]{};
		for (int j = 0; j < 2; j++)
		{
			targetColors[j] = cubeColors[layer][cube][const_edges[i].sides[j]];
		}
		if (targetColors[0] == colors[0] && targetColors[1] == colors[1] || targetColors[0] == colors[1] && targetColors[1] == colors[0])
		{
			return make_uint2(layer, cube);
		}
	}
	return make_uint2(100, 100);
}