#include "hip/hip_runtime.h"
#include "oll_edge_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ bool setOLLEdgeMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, bool match[4])
{
	for (int i = 0; i < 4; i++)
	{
		const Color color = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeReferences[i].layer][const_OLLEdgeReferences[i].cube][const_OLLEdgeReferences[i].side];
		const Color targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeTargetReferences[i].layer][const_OLLEdgeTargetReferences[i].cube][const_OLLEdgeTargetReferences[i].side];
		match[i] = color == targetColor;
	}
	int j = 0;
	while (j < 4 && !match[j])
	{
		j++;
	}
	return j < 4;
}

__device__ void setOLLEdgeNear(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, bool near[4])
{
	for (int i = 0; i < 4; i++)
	{
		const Color origin = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].origin.layer][const_OLLEdgeNearReferences[i].origin.cube][const_OLLEdgeNearReferences[i].origin.side];
		const Color leftTarget = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].leftTarget.layer][const_OLLEdgeNearReferences[i].leftTarget.cube][const_OLLEdgeNearReferences[i].leftTarget.side];
		const Color rightTarget = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].rightTarget.layer][const_OLLEdgeNearReferences[i].rightTarget.cube][const_OLLEdgeNearReferences[i].rightTarget.side];
		near[i] = origin == leftTarget || origin == rightTarget;
	}
}

__device__ void OLLEdgeSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Notation firstMove = None;
	bool match[4]{};
	int i = 0;
	while (!setOLLEdgeMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx, match))
	{
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
		i++;
	}
	switch (i)
	{
	case 1:
		firstMove = U;
		break;
	case 2:
		firstMove = U2;
		break;
	case 3:
		firstMove = Up;
		break;
	}
	
	dev_OLLEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][0] = firstMove;
	bool near[4]{};
	setOLLEdgeNear(cubeIdx, crossIdx, cornerIdx, edgeIdx, near);
	const Notation* sequence = findOLLEdgeSequence(match, near);
	executeOLLEdgeSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence);
}

__global__ void solveOLLEdgeThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	OLLEdgeSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solveOLLEdge()
{
	solveOLLEdgeThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}