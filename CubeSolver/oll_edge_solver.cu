#include "hip/hip_runtime.h"
#include "oll_edge_solver.cuh"

__device__ bool setOLLEdgeMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, bool match[4])
{
	for (int i = 0; i < 4; i++)
	{
		Color color = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeReferences[i].layer][const_OLLEdgeReferences[i].cube][const_OLLEdgeReferences[i].side];
		Color targetColor = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeTargetReferences[i].layer][const_OLLEdgeTargetReferences[i].cube][const_OLLEdgeTargetReferences[i].side];
		match[i] = color == targetColor;
	}
	int j = 0;
	while (j < 4 && !match[j])
	{
		j++;
	}
	return j < 4;
}

__device__ void setOLLEdgeNear(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, bool near[4])
{
	for (int i = 0; i < 4; i++)
	{
		Color origin = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].origin.layer][const_OLLEdgeNearReferences[i].origin.cube][const_OLLEdgeNearReferences[i].origin.side];
		Color leftTarget = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].leftTarget.layer][const_OLLEdgeNearReferences[i].leftTarget.cube][const_OLLEdgeNearReferences[i].leftTarget.side];
		Color rightTarget = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLEdgeNearReferences[i].rightTarget.layer][const_OLLEdgeNearReferences[i].rightTarget.cube][const_OLLEdgeNearReferences[i].rightTarget.side];
		near[i] = origin == leftTarget || origin == rightTarget;
	}
}

__device__ void OLLEdgeSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Notation firstMove = None;
	bool match[4]{};
	int i = 0;
	while (!setOLLEdgeMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx, match))
	{
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
		i++;
	}
	switch (i)
	{
	case 1:
		firstMove = U;
		break;
	case 2:
		firstMove = U2;
		break;
	case 3:
		firstMove = Up;
		break;
	}
	
	dev_OLLEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][0] = firstMove;
	bool near[4]{};
	setOLLEdgeNear(cubeIdx, crossIdx, cornerIdx, edgeIdx, near);
	const Notation* sequence = findOLLEdgeSequence(match, near);
	executeOLLEdgeSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence);
}

__global__ void solveOLLEdgeThread()
{
	int cubeIdx = blockIdx.x;

	int crossIdxX = blockIdx.y;
	int crossIdxY = blockIdx.z;
	uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	int cornerIdxX = threadIdx.x % 4;
	int cornerIdxY = threadIdx.x / 4;
	uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	int edgeIdxX = threadIdx.y;
	int edgeIdxY = threadIdx.z;
	uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	OLLEdgeSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solveOLLEdge()
{
	solveOLLEdgeThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}