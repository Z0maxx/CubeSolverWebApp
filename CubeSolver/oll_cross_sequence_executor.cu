#include "hip/hip_runtime.h"
#include "oll_cross_sequence_executor.cuh"

__device__ void executeOLLCrossSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Notation sequence[], const int idx)
{
	for (int i = 0; i < 6; i++)
	{
		dev_OLLCrossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][i] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, move.cubeLayer, move.direction, move.twice);
	}
}