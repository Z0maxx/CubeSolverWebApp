#include "hip/hip_runtime.h"
#include "cube_rotations.cuh"

__shared__ Color shr_tempCubeColors[6][3][9][6];

__device__ void rotateLayerColors(const int cubeIdx, const CubeLayer cubeLayer, const Direction direction)
{
    memcpy(shr_tempCubeColors[cubeIdx], dev_cubeColors[cubeIdx], sizeof(dev_cubeColors[cubeIdx]));

    const int alwaysMoveIdx = cubeLayer / 3;
    const int directionIdx = direction % 2;

    for (int x = 0; x < 4; x++)
    {
        const Move cornerMove = const_layerCornerMoves[cubeLayer][directionIdx][x];
        
        for (int y = 0; y < 2; y++)
        {
            shr_tempCubeColors[cubeIdx][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_cubeColors[cubeIdx][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempCubeColors[cubeIdx][cornerMove.targetLayer][cornerMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_cubeColors[cubeIdx][cornerMove.originLayer][cornerMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 4; x++)
    {
        const Move edgeMove = const_layerEdgeMoves[cubeLayer][directionIdx][x];
        shr_tempCubeColors[cubeIdx][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_cubeColors[cubeIdx][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        for (int y = 0; y < 2; y++)
        {
            shr_tempCubeColors[cubeIdx][edgeMove.targetLayer][edgeMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_cubeColors[cubeIdx][edgeMove.originLayer][edgeMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    memcpy(dev_cubeColors[cubeIdx], shr_tempCubeColors[cubeIdx], sizeof(shr_tempCubeColors[cubeIdx]));
}

__device__ void rotateFaceColors(const int cubeIdx, const MoveIdx moveIdx)
{
    if (moveIdx == MoveIdx_None) return;

    const int alwaysMoveIdx = moveIdx / 2;

    for (int x = 0; x < 8; x++)
    {
        const Move cornerMove = const_faceCornerMoves[moveIdx][x];
        for (int y = 0; y < 2; y++)
        {
            shr_tempCubeColors[cubeIdx][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_cubeColors[cubeIdx][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempCubeColors[cubeIdx][cornerMove.targetLayer][cornerMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][y]] = dev_cubeColors[cubeIdx][cornerMove.originLayer][cornerMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 12; x++)
    {
        const Move edgeMove = const_faceEdgeMoves[moveIdx][x];
        if (x < 8)
        {
            shr_tempCubeColors[cubeIdx][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_cubeColors[cubeIdx][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        }
        else
        {
            for (int y = 0; y < 2; y++)
            {
                shr_tempCubeColors[cubeIdx][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[y]] = dev_cubeColors[cubeIdx][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[y]];
            }
        }

        for (int i = 0; i < 2; i++)
        {
            shr_tempCubeColors[cubeIdx][edgeMove.targetLayer][edgeMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][i]] = dev_cubeColors[cubeIdx][edgeMove.originLayer][edgeMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][i]];
        }
    }

    for (int x = 0; x < 6; x++)
    {
        const Move centerMove = const_faceCenterMoves[moveIdx][x];
        shr_tempCubeColors[cubeIdx][centerMove.targetLayer][centerMove.targetCube][centerMove.targetSides[0]] = dev_cubeColors[cubeIdx][centerMove.originLayer][centerMove.originCube][centerMove.originSides[0]];
    }

    memcpy(dev_cubeColors[cubeIdx], shr_tempCubeColors[cubeIdx], sizeof(shr_tempCubeColors[cubeIdx]));
}

__device__ void turnLayer(const int cubeIdx, const CubeLayer cubeLayer, const Direction direction, const bool twice) {
    rotateLayerColors(cubeIdx, cubeLayer, direction);
    if (twice)
    {
        rotateLayerColors(cubeIdx, cubeLayer, direction);
    }
}

__device__ void turnCube(const int cubeIdx, const Direction direction, bool twice) {
    MoveIdx moveIdx = const_cubeTurnMoveIdxs[direction];
    rotateFaceColors(cubeIdx, moveIdx);
    if (twice)
    {
        rotateFaceColors(cubeIdx, moveIdx);
    }
}