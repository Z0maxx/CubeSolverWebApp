#include "hip/hip_runtime.h"
#include "cube_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ void printCubes()
{
	for (int i = 0; i < 6; i++)
	{
		printCube(dev_cubeColors[i]);
	}
}

__global__ void startSolveThread()
{
	turnCube(1, Direction_Up, false);
	dev_cubeRotationMoves[1] = x;
	turnCube(2, Direction_Down, false);
	dev_cubeRotationMoves[2] = xp;
	turnCube(3, Direction_Down, true);
	dev_cubeRotationMoves[3] = x2;
	turnCube(4, Direction_Tilt_Left, false);
	dev_cubeRotationMoves[4] = zp;
	turnCube(5, Direction_Tilt_Right, false);
	dev_cubeRotationMoves[5] = z;
}

void startSolve()
{
	startSolveThread CUDA_KERNEL(1, 1)();
	hipDeviceSynchronize();
	
}

Color crossCubeColors[6][4][6][3][9][6];
Color edgeCubeColors[6][4][6][4][6][4][6][3][9][6];
Color cornerCubeColors[6][4][6][4][6][3][9][6];

void solve(const int cube[3][9][6])
{
	int cubes[6][3][9][6]{};
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			for (int k = 0; k < 9; k++)
			{
				for (int l = 0; l < 6; l++)
				{
					cubes[i][j][k][l] = cube[j][k][l];
				}
			}
		}
	}
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipMemcpyToSymbol(HIP_SYMBOL(dev_cubeColors), cubes, sizeof(cubes));
	startSolve();
	solveCross();
	solveF2LCorner();
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	solveF2LEdge();
	solveOLLCross();
	solveOLLEdge();
	solvePLLCycle();
	solvePLLOrient();
}