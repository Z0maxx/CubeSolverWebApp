#include "hip/hip_runtime.h"
#include "oll_cross_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ bool setOLLCrossMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor, bool match[4])
{
	for (int i = 0; i < 4; i++)
	{
		match[i] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLCrossReferences[i].layer][const_OLLCrossReferences[i].cube][const_OLLCrossReferences[i].side] == targetColor;
	}
	int j = 0;
	while (j < 4 && match[j])
	{
		j++;
	}
	return j == 4;
}
__device__ void OLLCrossSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	const Color targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLCrossTargetReference.layer][const_OLLCrossTargetReference.cube][const_OLLCrossTargetReference.side];
	bool match[4];
	int idx = 0;

	while (!setOLLCrossMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor, match))
	{
		const Notation* sequence = findOLLCrossSequence(match);
		executeOLLCrossSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence, idx);
		idx++;
	}
}

__global__ void solveOLLCrossThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	OLLCrossSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solveOLLCross()
{
	solveOLLCrossThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}