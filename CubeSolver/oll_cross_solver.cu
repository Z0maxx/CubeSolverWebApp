#include "hip/hip_runtime.h"
#include "oll_cross_solver.cuh"

__device__ bool setOLLCrossMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor, bool match[4])
{
	for (int i = 0; i < 4; i++)
	{
		match[i] = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLCrossReferences[i].layer][const_OLLCrossReferences[i].cube][const_OLLCrossReferences[i].side] == targetColor;
	}
	int j = 0;
	while (j < 4 && match[j])
	{
		j++;
	}
	return j == 4;
}
__device__ void OLLCrossSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Color targetColor = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_OLLCrossTargetReference.layer][const_OLLCrossTargetReference.cube][const_OLLCrossTargetReference.side];
	bool match[4];
	int idx = 0;
	bool foundSequence = true;

	while (foundSequence && !setOLLCrossMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor, match))
	{
		const Notation* sequence = findOLLCrossSequence(match);
		if (sequence == 0)
		{
			edgeError = true;
			foundSequence = false;
		}
		else
		{
			executeOLLCrossSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence, idx);
			idx++;
		}
	}
}

__global__ void solveOLLCrossThread()
{
	int cubeIdx = blockIdx.x;

	int crossIdxX = blockIdx.y;
	int crossIdxY = blockIdx.z;
	uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	int cornerIdxX = threadIdx.x % 4;
	int cornerIdxY = threadIdx.x / 4;
	uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	int edgeIdxX = threadIdx.y;
	int edgeIdxY = threadIdx.z;
	uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	OLLCrossSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solveOLLCross()
{
	solveOLLCrossThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}