#include "hip/hip_runtime.h"
#include "f2l_edge_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ int dev_F2LEdgeRotations[6][4][6][4][6][4][6];

__device__ void solveF2LEdgePiece(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const int idx, const int solveOrderIdx)
{
	SolveRotation solveRotation = findSolveRotation(dev_F2LEdgeRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], solveOrderIdx);
	if (solveRotation.rotation != 99)
	{
		turnF2LEdgeCube(cubeIdx, crossIdx, cornerIdx, edgeIdx, solveRotation.direction, solveRotation.twice);
		dev_F2LEdgeRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx] = solveRotation.notation;
		dev_F2LEdgeRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y] = solveRotation.newRotation;
	}
	
	Color colors[2]{};
	for (int i = 0; i < 2; i++)
	{
		colors[i] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_F2LEdgeReferences[i].layer][const_F2LEdgeReferences[i].cube][const_F2LEdgeReferences[i].side];
	}
	const uint2 edge = findEdge(dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], colors, 8);
	const Color targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_F2LEdgeTargetReference.layer][const_F2LEdgeTargetReference.cube][const_F2LEdgeTargetReference.side];
	const Notation* sequence = findF2LEdgeSequence(edge, dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edge.x][edge.y], targetColor);
	executeF2LEdgeSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence, idx);
}

__global__ void solveF2LEdgeThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	for (int i = 0; i < 4; i++)
	{
		solveF2LEdgePiece(cubeIdx, crossIdx, cornerIdx, edgeIdx, i, solveOrders[edgeIdx.x][edgeIdx.y][i]);
	}
}

__global__ void copyF2LEdgeColorThread()
{
	for (int i = 0; i < 6; i++)
	{
		memcpy(dev_F2LEdgeCubeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y][threadIdx.z][i], dev_F2LCornerCubeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y], sizeof(dev_F2LCornerCubeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y]));
	}
}

void solveF2LEdge()
{
	copyF2LEdgeColorThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6, 4))();
	hipDeviceSynchronize();
	solveF2LEdgeThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}