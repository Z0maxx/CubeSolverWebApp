#include "hip/hip_runtime.h"
#include "f2l_edge_solver.cuh"

__device__ int dev_F2LEdgeRotations[6][4][6][4][6][4][6];

__device__ void solveF2LEdgePiece(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const int idx, const int solveOrderIdx)
{
	SolveRotation solveRotation = findSolveRotation(dev_F2LEdgeRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], solveOrderIdx);
	if (solveRotation.rotation != 99)
	{
		turnF2LEdgeCube(cubeIdx, crossIdx, cornerIdx, edgeIdx, solveRotation.direction, solveRotation.twice);
		dev_F2LEdgeRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx] = solveRotation.notation;
		dev_F2LEdgeRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y] = solveRotation.newRotation;
	}
	
	Color colors[2]{};
	for (int i = 0; i < 2; i++)
	{
		colors[i] = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_F2LEdgeReferences[i].layer][const_F2LEdgeReferences[i].cube][const_F2LEdgeReferences[i].side];
	}
	uint2 edge = findEdge(dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], colors, 8);
	Color targetColor = dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_F2LEdgeTargetReference.layer][const_F2LEdgeTargetReference.cube][const_F2LEdgeTargetReference.side];
	const Notation* sequence = findF2LEdgeSequence(edge, dev_F2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edge.x][edge.y], targetColor);
	executeF2LEdgeSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence, idx);
}

__global__ void solveF2LEdgeThread()
{
	int cubeIdx = blockIdx.x;

	int crossIdxX = blockIdx.y;
	int crossIdxY = blockIdx.z;
	uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	int cornerIdxX = threadIdx.x % 4;
	int cornerIdxY = threadIdx.x / 4;
	uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	int edgeIdxX = threadIdx.y;
	int edgeIdxY = threadIdx.z;
	uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	for (int i = 0; i < 4; i++)
	{
		solveF2LEdgePiece(cubeIdx, crossIdx, cornerIdx, edgeIdx, i, const_solveOrders[edgeIdx.x][edgeIdx.y][i]);
	}
}

__global__ void copyF2LEdgeColorThread()
{
	for (int i = 0; i < 6; i++)
	{
		memcpy(dev_F2LEdgeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y][threadIdx.z][i], dev_F2LCornerColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y], sizeof(dev_F2LCornerColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y]));
		memcpy(dev_tempF2LEdgeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y][threadIdx.z][i], dev_F2LCornerColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y], sizeof(dev_F2LCornerColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y]));

	}
}

void solveF2LEdge()
{
	copyF2LEdgeColorThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6, 4))();
	hipDeviceSynchronize();
	solveF2LEdgeThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}