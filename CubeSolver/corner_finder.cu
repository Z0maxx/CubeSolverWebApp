#include "hip/hip_runtime.h"
#include "corner_finder.cuh"

__constant__ const Corner const_corners[8] =
{
	{
		Layer_Front,
		Cube_Top_Left,
		{ Side_Left, Side_Front, Side_Top }
	},
	{
		Layer_Front,
		Cube_Top_Right,
		{ Side_Right, Side_Front, Side_Top }
	},
	{
		Layer_Back,
		Cube_Top_Left,
		{ Side_Left, Side_Back, Side_Top }
	},
	{
		Layer_Back,
		Cube_Top_Right,
		{ Side_Right, Side_Back, Side_Top }
	},
	{
		Layer_Front,
		Cube_Bottom_Left,
		{ Side_Left, Side_Front, Side_Bottom }
	},
	{
		Layer_Front,
		Cube_Bottom_Right,
		{ Side_Right, Side_Front, Side_Bottom }
	},
	{
		Layer_Back,
		Cube_Bottom_Left,
		{ Side_Left, Side_Back, Side_Bottom }
	},
	{
		Layer_Back,
		Cube_Bottom_Right,
		{ Side_Right, Side_Back, Side_Bottom }
	}
};

__device__ const uint2 findCorner(const Color cubeColors[3][9][6], const Color colors[3], const int count)
{
	for (int i = 0; i < count; i++)
	{
		const Layer layer = const_corners[i].layer;
		const Cube cube = const_corners[i].cube;
		Color targetColors[3]{};
		for (int j = 0; j < 3; j++)
		{
			targetColors[j] = cubeColors[layer][cube][const_corners[i].sides[j]];
		}
		if (targetColors[0] == colors[0] && targetColors[1] == colors[1] && targetColors[2] == colors[2] ||
			targetColors[0] == colors[0] && targetColors[1] == colors[2] && targetColors[2] == colors[1] ||
			targetColors[0] == colors[1] && targetColors[1] == colors[0] && targetColors[2] == colors[2] ||
			targetColors[0] == colors[1] && targetColors[1] == colors[2] && targetColors[2] == colors[0] ||
			targetColors[0] == colors[2] && targetColors[1] == colors[0] && targetColors[2] == colors[1] ||
			targetColors[0] == colors[2] && targetColors[1] == colors[1] && targetColors[2] == colors[0])
		{
			return make_uint2(layer, cube);
		}
	}
	return make_uint2(100, 100);
}