#include "hip/hip_runtime.h"
#include "variables.cuh"
#include "cross_rotations.cuh"

__shared__ Color shr_tempCrossColors[4][6][3][9][6];

__device__ void rotateCrossLayerColors(const int cubeIdx, const uint2 crossIdx, const CubeLayer cubeLayer, const Direction direction)
{
    memcpy(shr_tempCrossColors[crossIdx.x][crossIdx.y], dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y], sizeof(shr_tempCrossColors[crossIdx.x][crossIdx.y]));

    int alwaysMoveIdx = cubeLayer / 3;
    int directionIdx = direction % 2;

    for (int x = 0; x < 4; x++)
    {
        Move cornerMove = const_layerCornerMoves[cubeLayer][directionIdx][x];

        for (int y = 0; y < 2; y++)
        {
            shr_tempCrossColors[crossIdx.x][crossIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempCrossColors[crossIdx.x][crossIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][cornerMove.originLayer][cornerMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 4; x++)
    {
        Move edgeMove = const_layerEdgeMoves[cubeLayer][directionIdx][x];
        shr_tempCrossColors[crossIdx.x][crossIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        for (int y = 0; y < 2; y++)
        {
            shr_tempCrossColors[crossIdx.x][crossIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][edgeMove.originLayer][edgeMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    memcpy(dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y], shr_tempCrossColors[crossIdx.x][crossIdx.y], sizeof(shr_tempCrossColors[crossIdx.x][crossIdx.y]));
}

__device__ void rotateCrossFaceColors(const int cubeIdx, const uint2 crossIdx, const MoveIdx moveIdx)
{
    if (moveIdx == MoveIdx_None) return;

    int alwaysMoveIdx = moveIdx / 2;

    for (int x = 0; x < 8; x++)
    {
        Move cornerMove = const_faceCornerMoves[moveIdx][x];
        for (int y = 0; y < 2; y++)
        {
            shr_tempCrossColors[crossIdx.x][crossIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempCrossColors[crossIdx.x][crossIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][cornerMove.originLayer][cornerMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 12; x++)
    {
        Move edgeMove = const_faceEdgeMoves[moveIdx][x];
        if (x < 8)
        {
            shr_tempCrossColors[crossIdx.x][crossIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        }
        else
        {
            for (int y = 0; y < 2; y++)
            {
                shr_tempCrossColors[crossIdx.x][crossIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[y]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[y]];
            }
        }

        for (int i = 0; i < 2; i++)
        {
            shr_tempCrossColors[crossIdx.x][crossIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][i]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][edgeMove.originLayer][edgeMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][i]];
        }
    }

    for (int x = 0; x < 6; x++)
    {
        Move centerMove = const_faceCenterMoves[moveIdx][x];
        shr_tempCrossColors[crossIdx.x][crossIdx.y][centerMove.targetLayer][centerMove.targetCube][centerMove.targetSides[0]] = dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y][centerMove.originLayer][centerMove.originCube][centerMove.originSides[0]];
    }

    memcpy(dev_crossColors[cubeIdx][crossIdx.x][crossIdx.y], shr_tempCrossColors[crossIdx.x][crossIdx.y], sizeof(shr_tempCrossColors[crossIdx.x][crossIdx.y]));
}

__device__ void turnCrossLayer(const int cubeIdx, const uint2 crossIdx, const CubeLayer cubeLayer, const Direction direction, const bool twice) {
    rotateCrossLayerColors(cubeIdx, crossIdx, cubeLayer, direction);
    if (twice)
    {
        rotateCrossLayerColors(cubeIdx, crossIdx, cubeLayer, direction);
    }
}

__device__ void turnCrossCube(const int cubeIdx, const uint2 crossIdx, const Direction direction, const bool twice) {
    MoveIdx moveIdx = const_cubeTurnMoveIdxs[direction];
    rotateCrossFaceColors(cubeIdx, crossIdx, moveIdx);
    if (twice)
    {
        rotateCrossFaceColors(cubeIdx, crossIdx, moveIdx);
    }
}