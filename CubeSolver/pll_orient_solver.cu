#include "hip/hip_runtime.h"
#include "pll_orient_solver.cuh"

__device__ bool doesPLLOrientEdgeMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Color colors[2]{};
	for (int i = 0; i < 2; i++)
	{
		colors[i] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientEdgeMatchReferences[i].layer][const_PLLOrientEdgeMatchReferences[i].cube][const_PLLOrientEdgeMatchReferences[i].side];
	}
	return colors[0] == colors[1];
}

__device__ bool doesPLLOrientCornerMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Color colors[4]{};
	for (int i = 0; i < 4; i++)
	{
		colors[i] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientCornerMatchReferences[i].layer][const_PLLOrientCornerMatchReferences[i].cube][const_PLLOrientCornerMatchReferences[i].side];
	}
	return colors[0] == colors[1] && colors[2] == colors[3];
}

__device__ int PLLOrientCorrectCount(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	int j = 0;
	while (j < 4 && dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientTargetReferences[j].layer][const_PLLOrientTargetReferences[j].cube][const_PLLOrientTargetReferences[j].side] == targetColor)
	{
		j++;
	}
	return j;
}

__device__ bool isPLLOrientCurrentCorrect(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	return dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientCurrentTargetReference.layer][const_PLLOrientCurrentTargetReference.cube][const_PLLOrientCurrentTargetReference.side] == targetColor;
}

__device__ bool isPLLOrientTopLayerCorrect(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	return dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientFinalReferences[0].layer][const_PLLOrientFinalReferences[0].cube][const_PLLOrientFinalReferences[0].side] == targetColor;
}

__device__ void PLLOrientSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Color targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientReference.layer][const_PLLOrientReference.cube][const_PLLOrientReference.side];
	int idx = 0;
	int roundIdx = 0;
	int correctCount = PLLOrientCorrectCount(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
	bool edgeMatch = doesPLLOrientEdgeMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
	bool cornerMatch = doesPLLOrientCornerMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
	while (correctCount != 4 && !(edgeMatch && cornerMatch && correctCount == 3))
	{
		bool currentCorrect = isPLLOrientCurrentCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
		roundIdx = 0;

		while (!currentCorrect && !(edgeMatch && cornerMatch && correctCount == 3))
		{
			executePLLOrientSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, idx, roundIdx);
			currentCorrect = isPLLOrientCurrentCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
			if (!currentCorrect)
			{
				roundIdx++;
			}

			edgeMatch = doesPLLOrientEdgeMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
			cornerMatch = doesPLLOrientCornerMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
			if (edgeMatch && cornerMatch && !currentCorrect && correctCount == 3)
			{
				cornerError = true;
			}
		}
		correctCount = PLLOrientCorrectCount(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
		if (correctCount != 4)
		{
			turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
			dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][roundIdx][4] = U;
			idx++;
		}

		edgeMatch = doesPLLOrientEdgeMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
		cornerMatch = doesPLLOrientCornerMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx);
		if (edgeMatch && cornerMatch && correctCount == 3)
		{
			cornerError = true;
		}
	}
	int i = 0;
	targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientFinalReferences[1].layer][const_PLLOrientFinalReferences[1].cube][const_PLLOrientFinalReferences[1].side];
	while (!isPLLOrientTopLayerCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor))
	{
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
		i++;
	}

	Notation lastMove = None;
	switch (i)
	{
	case 1:
		lastMove = U;
		break;
	case 2:
		lastMove = U2;
		break;
	case 3:
		lastMove = Up;
		break;
	}

	dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][roundIdx][4] = lastMove;
}

__global__ void solvePLLOrientThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	PLLOrientSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solvePLLOrient()
{
	solvePLLOrientThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}