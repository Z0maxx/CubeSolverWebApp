#include "hip/hip_runtime.h"
#include "pll_orient_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ bool isPLLOrientCorrect(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	int j = 0;
	while (j < 4 && dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientTargetReferences[j].layer][const_PLLOrientTargetReferences[j].cube][const_PLLOrientTargetReferences[j].side] == targetColor)
	{
		j++;
	}
	return j == 4;
}

__device__ bool isPLLOrientCurrentCorrect(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	return dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientCurrentTargetReference.layer][const_PLLOrientCurrentTargetReference.cube][const_PLLOrientCurrentTargetReference.side] == targetColor;
}

__device__ bool isPLLOrientTopLayerCorrect(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Color targetColor)
{
	return dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientFinalReferences[0].layer][const_PLLOrientFinalReferences[0].cube][const_PLLOrientFinalReferences[0].side] == targetColor;
}

__device__ void PLLOrientSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	Color targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientReference.layer][const_PLLOrientReference.cube][const_PLLOrientReference.side];
	int idx = 0;
	int roundIdx = 0;
	bool correct = isPLLOrientCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
	while (!correct)
	{
		bool currentCorrect = isPLLOrientCurrentCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
		roundIdx = 0;
		while (!currentCorrect)
		{
			executePLLOrientSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, idx, roundIdx);
			currentCorrect = isPLLOrientCurrentCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
			if (!currentCorrect)
			{
				roundIdx++;
			}

		}
		correct = isPLLOrientCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor);
		if (!correct)
		{
			turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
			dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][roundIdx][4] = U;
			idx++;
		}
	}
	int i = 0;
	targetColor = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLOrientFinalReferences[1].layer][const_PLLOrientFinalReferences[1].cube][const_PLLOrientFinalReferences[1].side];
	while (!isPLLOrientTopLayerCorrect(cubeIdx, crossIdx, cornerIdx, edgeIdx, targetColor))
	{
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, CubeLayer_Top, Direction_Left, false);
		i++;
	}

	Notation lastMove = None;
	switch (i)
	{
	case 1:
		lastMove = U;
		break;
	case 2:
		lastMove = U2;
		break;
	case 3:
		lastMove = Up;
		break;
	}

	dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][roundIdx][4] = lastMove;
}

__global__ void solvePLLOrientThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	PLLOrientSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solvePLLOrient()
{
	solvePLLOrientThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}