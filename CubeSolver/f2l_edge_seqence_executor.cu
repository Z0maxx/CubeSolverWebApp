#include "hip/hip_runtime.h"
#include "f2l_edge_sequence_executor.cuh"

__device__ void executeF2LEdgeSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Notation sequence[], const int idx)
{
	int i = 0;
	while (i < 16 && sequence[i] != None)
	{
		dev_F2LEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][idx][i] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, move.cubeLayer, move.direction, move.twice);
		i++;
	}
}