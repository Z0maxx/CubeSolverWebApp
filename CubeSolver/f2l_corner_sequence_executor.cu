#include "hip/hip_runtime.h"
#include "f2l_corner_sequence_executor.cuh"

__device__ void executeF2LCornerSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const Notation sequence[], const int idx)
{
	int i = 0;
	while (i < 8 && sequence[i] != None)
	{
		dev_F2LCornerLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][idx][i] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnF2LCornerLayer(cubeIdx, crossIdx, cornerIdx, move.cubeLayer, move.direction, move.twice);
		i++;
	}
}