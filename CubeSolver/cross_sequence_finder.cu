#include "hip/hip_runtime.h"
#include "cross_sequence_finder.cuh"

__device__ const Notation* findCrossSequence(const uint2 edge, const Color colors[6], const Color targetColor)
{
	for (int i = 0; i < 24; i++)
	{
		if (const_crossSequences[i].layer == edge.x && const_crossSequences[i].cube == edge.y && colors[const_crossSequences[i].side] == targetColor)
		{
			return const_crossSequences[i].sequence;
		}
	}
	return { 0 };
}