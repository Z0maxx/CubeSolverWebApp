#include "error_printer.cuh"

void printError(hipError_t error)
{
	printf("%s %s\n", hipGetErrorName(error), hipGetErrorString(error));
}