#include "hip/hip_runtime.h"
#include "cross_sequence_executor.cuh"

__device__ void executeCrossSequence(const int cubeIdx, const uint2 crossIdx, const Notation sequence[], const int idx)
{
	int i = 0;
	while (i < 4 && sequence[i] != None)
	{
		dev_crossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][idx][i] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnCrossLayer(cubeIdx, crossIdx, move.cubeLayer, move.direction, move.twice);
		i++;
	}
}