﻿#include "cube_solver.cuh"

int main(int argc, char* argv[])
{
    int idx = 1;
    int colors[3][9][6] = {};
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 9; j++)
        {
            for (int k = 0; k < 6; k++)
            {
                const char *a = argv[idx];
                colors[i][j][k] = atoi(a);
                idx++;
            }
        }
    }
    hipSetDeviceFlags(hipDeviceScheduleSpin);
    solve(colors);
    findSequence();
    Notation sequence[263]{};
    hipMemcpyFromSymbol(sequence, HIP_SYMBOL(dev_sequence), sizeof(sequence));
    for (int i = 0; i < 263; i++)
    {
        if (sequence[i] != None)
        {
            const TranslatedNotation move = translateNotation(sequence[i]);
            printf("%d ", sequence[i]);
        }
    }
}
