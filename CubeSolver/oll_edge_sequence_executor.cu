#include "hip/hip_runtime.h"
#include "oll_edge_sequence_executor.cuh"

__device__ void executeOLLEdgeSequence(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Notation sequence[])
{
	int i = 0;
	while (i < 15 && sequence[i] != None)
	{
		dev_OLLEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i + 1] = sequence[i];
		const TranslatedNotation move = translateNotation(sequence[i]);
		turnF2LEdgeLayer(cubeIdx, crossIdx, cornerIdx, edgeIdx, move.cubeLayer, move.direction, move.twice);
		i++;
	}
}