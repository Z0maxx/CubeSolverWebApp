#include "hip/hip_runtime.h"
#include "variables.cuh"
#include "f2l_edge_rotations.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ Color dev_tempF2LEdgeColors[6][4][6][4][6][4][6][3][9][6];

__device__ void rotateF2LEdgeLayerColors(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const CubeLayer cubeLayer, const Direction direction)
{
    memcpy(dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], sizeof(dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y]));

    const int alwaysMoveIdx = cubeLayer / 3;
    const int directionIdx = direction % 2;

    for (int x = 0; x < 4; x++)
    {
        const Move cornerMove = const_layerCornerMoves[cubeLayer][directionIdx][x];

        for (int y = 0; y < 2; y++)
        {
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.originLayer][cornerMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 4; x++)
    {
        const Move edgeMove = const_layerEdgeMoves[cubeLayer][directionIdx][x];
        dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        for (int y = 0; y < 2; y++)
        {
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.originLayer][edgeMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    memcpy(dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], sizeof(dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y]));
}

__device__ void rotateF2LEdgeFaceColors(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const MoveIdx moveIdx)
{
    if (moveIdx == MoveIdx_None) return;

    const int alwaysMoveIdx = moveIdx / 2;

    for (int x = 0; x < 8; x++)
    {
        const Move cornerMove = const_faceCornerMoves[moveIdx][x];
        for (int y = 0; y < 2; y++)
        {
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][cornerMove.originLayer][cornerMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 12; x++)
    {
        const Move edgeMove = const_faceEdgeMoves[moveIdx][x];
        if (x < 8)
        {
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        }
        else
        {
            for (int y = 0; y < 2; y++)
            {
                dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[y]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[y]];
            }
        }

        for (int i = 0; i < 2; i++)
        {
            dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][i]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][edgeMove.originLayer][edgeMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][i]];
        }
    }

    for (int x = 0; x < 6; x++)
    {
        const Move centerMove = const_faceCenterMoves[moveIdx][x];
        dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][centerMove.targetLayer][centerMove.targetCube][centerMove.targetSides[0]] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][centerMove.originLayer][centerMove.originCube][centerMove.originSides[0]];
    }

    memcpy(dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], sizeof(dev_tempF2LEdgeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y]));
}

__device__ void turnF2LEdgeLayer(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const CubeLayer cubeLayer, const Direction direction, const bool twice) {
    rotateF2LEdgeLayerColors(cubeIdx, crossIdx, cornerIdx, edgeIdx, cubeLayer, direction);
    if (twice)
    {
        rotateF2LEdgeLayerColors(cubeIdx, crossIdx, cornerIdx, edgeIdx, cubeLayer, direction);
    }
}

__device__ void turnF2LEdgeCube(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, const Direction direction, const bool twice) {
    MoveIdx moveWithLayerIdx = const_cubeTurnMoveIdxs[direction];
    rotateF2LEdgeFaceColors(cubeIdx, crossIdx, cornerIdx, edgeIdx, moveWithLayerIdx);
    if (twice)
    {
        rotateF2LEdgeFaceColors(cubeIdx, crossIdx, cornerIdx, edgeIdx, moveWithLayerIdx);
    }
}