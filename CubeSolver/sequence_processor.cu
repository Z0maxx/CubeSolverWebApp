#include "hip/hip_runtime.h"
#include "sequence_processor.cuh"
#include <stdio.h>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ int dev_movesCountIdx[solveCount];
__device__ int dev_movesCount[solveCount];

__device__ int dev_tempMovesCount[solveCount];
__device__ int dev_tempMovesCountIdx[solveCount];

__global__ void setMovesThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	int idx = cubeIdx * 13824 + crossIdx.x * 3456 + crossIdx.y * 576 + cornerIdx.x * 144 + cornerIdx.y * 24 + edgeIdx.x * 6 + edgeIdx.y;
	dev_movesCountIdx[idx] = idx;

	dev_moves[idx][0] = dev_cubeRotationMoves[cubeIdx];
	int pos = 1;
	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_crossRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][i];
		pos++;
		for (int j = 0; j < 4; j++)
		{
			dev_moves[idx][pos] = dev_crossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_F2LCornerRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][i];
		pos++;
		for (int j = 0; j < 8; j++)
		{
			dev_moves[idx][pos] = dev_F2LCornerLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_F2LEdgeRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i];
		pos++;
		for (int j = 0; j < 16; j++)
		{
			dev_moves[idx][pos] = dev_F2LEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_moves[idx][pos] = dev_OLLCrossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 16; i++)
	{
		dev_moves[idx][pos] = dev_OLLEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i];
		pos++;
	}

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			dev_moves[idx][pos] = dev_PLLCycleLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			for (int k = 0; k < 5; k++)
			{
				dev_moves[idx][pos] = dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j][k];
				pos++;
			}
		}
	}
}

void setMoves()
{
	setMovesThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}

__global__ void setMovesCountThread(bool whiteOnly)
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	const int idx = cubeIdx * 13824 + crossIdx.x * 3456 + crossIdx.y * 576 + cornerIdx.x * 144 + cornerIdx.y * 24 + edgeIdx.x * 6 + edgeIdx.y;

	int count = 0;
	if (whiteOnly && dev_F2LEdgeCubeColors[cubeIdx][0][0][0][0][0][0][Layer_Middle][Cube_Bottom][Side_Bottom] != White)
	{
		count = 9999;
	}
	else
	{
		Notation last = None;
		int lastIdx = -1;

		for (int i = 0; i < maxStepCount; i++)
		{
			Notation current = dev_moves[idx][i];
			if (current != None)
			{
				if (lastIdx != -1)
				{
					//U + U' = None and U2 + U2 = None
					if (abs(current - last) == 1 && (current % 3 == 1 && last % 3 == 2 || current % 3 == 2 && last % 3 == 1) ||
						current == last && current % 3 == 0)
					{
						dev_moves[idx][i] = None;
						dev_moves[idx][lastIdx] = None;
						count--;
					}

					//U + U = U2 and U' + U' = U2
					else if (current == last && current % 3 != 0)
					{
						int newMove = current + 2;
						if (current % 3 == 2)
						{
							newMove = current + 1;
						}
						dev_moves[idx][i] = (Notation)newMove;
						dev_moves[idx][lastIdx] = None;
					}

					//U + U2 = U'
					else if (abs(current - last) == 2 && (current % 3 == 0 && last % 3 == 1 || current % 3 == 1 && last % 3 == 0))
					{
						dev_moves[idx][i] = (Notation)((current < last ? current : last) + 1);
						dev_moves[idx][lastIdx] = None;
					}

					//U' + U2 = U
					else if (abs(current - last) == 1 && (current % 3 == 0 && last % 3 == 2 || current % 3 == 2 && last % 3 == 0))
					{
						dev_moves[idx][i] = (Notation)((current < last ? current : last) - 1);
						dev_moves[idx][lastIdx] = None;
					}

					else {
						count++;
					}
				}
				else
				{
					count++;
				}

				if (dev_moves[idx][i] != None)
				{
					last = dev_moves[idx][i];
					lastIdx = i;
				}
				else if (lastIdx != -1 && dev_moves[idx][lastIdx] == None)
				{
					int j = lastIdx;
					while (j > -1 && dev_moves[idx][j] == None)
					{
						j--;
					}
					if (j != -1)
					{
						last = dev_moves[idx][j];
					}
					lastIdx = j;
				}
			}
		}
	}

	dev_movesCount[idx] = count;
}

void setMovesCount(bool whiteOnly)
{
	setMovesCountThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))(whiteOnly);
	hipDeviceSynchronize();
}

__device__ void setSequenceInner(int needed, int idx, int idx1, int idx2, int* dev_fromMovesCount, int* dev_toMovesCount, int* dev_fromMovesCountIdx, int* dev_toMovesCountIdx)
{
	int a = dev_fromMovesCount[idx1];
	int b = dev_fromMovesCount[idx2];
	bool first = a < b;
	int movesCountIdx = first ? dev_fromMovesCountIdx[idx1] : dev_fromMovesCountIdx[idx2];
	dev_toMovesCount[idx] = first ? a : b;
	dev_toMovesCountIdx[idx] = movesCountIdx;
	if (needed == 1)
	{
		memcpy(dev_sequence, dev_moves[movesCountIdx], sizeof(dev_sequence));
	}
}

__global__ void setSequenceThread(const int needed, bool fromTemp)
{
	int idx = blockIdx.x * 1024 + threadIdx.x;
	if (idx < needed)
	{
		int idx1 = idx * 2;
		int idx2 = idx1 + 1;
		if (fromTemp)
		{
			setSequenceInner(needed, idx, idx1, idx2, dev_tempMovesCount, dev_movesCount, dev_tempMovesCountIdx, dev_movesCountIdx);
		}
		else
		{
			setSequenceInner(needed, idx, idx1, idx2, dev_movesCount, dev_tempMovesCount, dev_movesCountIdx, dev_tempMovesCountIdx);
		}
	}
}

void setSequence()
{
	int needed = 41472;
	bool fromTemp = false;
	while (needed != 0)
	{
		setSequenceThread CUDA_KERNEL((int)ceil((double)needed / 1024), needed > 1024 ? 1024 : needed)(needed, fromTemp);
		hipDeviceSynchronize();
		if (needed > 1)
		{
			needed = (int)ceil((double)needed / 2);
		}
		else
		{
			needed = 0;
		}
		fromTemp = !fromTemp;
	}
}

void findSequence(bool whiteOnly)
{
	setMoves();
	setMovesCount(whiteOnly);
	setSequence();
}