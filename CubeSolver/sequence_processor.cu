#include "hip/hip_runtime.h"
#include "sequence_processor.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ int dev_movesCountIdx[82944];
__device__ int dev_movesCount[82944];

__global__ void setMovesThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	int idx = cubeIdx * 13824 + crossIdx.x * 3456 + crossIdx.y * 576 + cornerIdx.x * 144 + cornerIdx.y * 24 + edgeIdx.x * 6 + edgeIdx.y;
	dev_movesCountIdx[idx] = idx;

	dev_moves[idx][0] = dev_cubeRotationMoves[cubeIdx];
	int pos = 1;
	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_crossRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][i];
		pos++;
		for (int j = 0; j < 4; j++)
		{
			dev_moves[idx][pos] = dev_crossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_F2LCornerRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][i];
		pos++;
		for (int j = 0; j < 8; j++)
		{
			dev_moves[idx][pos] = dev_F2LCornerLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		dev_moves[idx][pos] = dev_F2LEdgeRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i];
		pos++;
		for (int j = 0; j < 16; j++)
		{
			dev_moves[idx][pos] = dev_F2LEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_moves[idx][pos] = dev_OLLCrossLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 16; i++)
	{
		dev_moves[idx][pos] = dev_OLLEdgeLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i];
		pos++;
	}

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			dev_moves[idx][pos] = dev_PLLCycleLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j];
			pos++;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			for (int k = 0; k < 5; k++)
			{
				dev_moves[idx][pos] = dev_PLLOrientLayerMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][i][j][k];
				pos++;
			}
		}
	}
}

void setMoves()
{
	setMovesThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}

__global__ void setMovesCountThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	const int idx = cubeIdx * 13824 + crossIdx.x * 3456 + crossIdx.y * 576 + cornerIdx.x * 144 + cornerIdx.y * 24 + edgeIdx.x * 6 + edgeIdx.y;

	int count = 0;
	Notation last = None;
	int lastIdx = -1;

	for (int i = 0; i < 263; i++)
	{
		Notation current = dev_moves[idx][i];
		if (current != None)
		{
			if (lastIdx != -1)
			{
				if (abs(current - last) == 1 && (current % 3 == 1 && last % 3 == 2 || current % 3 == 2 && last % 3 == 1) ||
					current == last && current % 3 == 0)
				{
					dev_moves[idx][i] = None;
					dev_moves[idx][lastIdx] = None;
					count--;
				}

				else if (current == last && current % 3 != 0)
				{
					int newMove = current + 2;
					if (current % 3 == 2)
					{
						newMove = current + 1;
					}
					dev_moves[idx][i] = (Notation)newMove;
					dev_moves[idx][lastIdx] = None;
				}

				else if (abs(current - last) == 2 && (current % 3 == 0 && last % 3 == 1 || current % 3 == 1 && last % 3 == 0))
				{
					dev_moves[idx][i] = (Notation)((current < last ? current : last) + 1);
					dev_moves[idx][lastIdx] = None;
				}

				else if (abs(current - last) == 1 && (current % 3 == 0 && last % 3 == 2 || current % 3 == 2 && last % 3 == 0))
				{
					dev_moves[idx][i] = (Notation)((current < last ? current : last) - 1);
					dev_moves[idx][lastIdx] = None;
				}

				else {
					count++;
				}
			}
			else
			{
				count++;
			}

			if (dev_moves[idx][i] != None)
			{
				last = dev_moves[idx][i];
				lastIdx = i;
			}
			else if (lastIdx != -1)
			{
				if (dev_moves[idx][lastIdx] == None)
				{
					int j = lastIdx;
					while (j > -1 && dev_moves[idx][j] == None)
					{
						j--;
					}
					if (j != -1)
					{
						last = dev_moves[idx][j];
					}
					lastIdx = j;
				}
			}
		}

	}
	dev_movesCount[idx] = count;
}

void setMovesCount()
{
	setMovesCountThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}

__global__ void setSequenceThread(const int needed)
{
	const int idx = blockIdx.x * 1024 + threadIdx.x;
	if (idx < needed)
	{
		const int idx1 = idx * 2;
		const int idx2 = idx1 + 1;
		const int a = dev_movesCount[idx1];
		const int b = dev_movesCount[idx2];
		const bool first = a < b;
		dev_movesCount[idx] = first ? a : b;
		dev_movesCountIdx[idx] = first ? dev_movesCountIdx[idx1] : dev_movesCountIdx[idx2];
	}
	if (needed == 1)
	{
		memcpy(dev_sequence, dev_moves[dev_movesCountIdx[0]], sizeof(dev_sequence));
	}
}

void setSequence()
{
	int needed = 41472;
	while (needed != 0)
	{
		setSequenceThread CUDA_KERNEL(ceil((double)needed / 1024), needed > 1024 ? 1024 : needed)(needed);
		hipDeviceSynchronize();
		if (needed > 1)
		{
			needed = ceil((double)needed / 2);
		}
		else
		{
			needed = 0;
		}
	}
}
void findSequence()
{
	setMoves();
	setMovesCount();
	setSequence();
}