#include "hip/hip_runtime.h"
#include "pll_cycle_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ bool setPLLCycleMatch(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx, bool match[4])
{
	for (int i = 0; i < 4; i++)
	{
		Color colors[3]{};
		for (int j = 0; j < 3; j++)
		{
			colors[j] = dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y][const_PLLCycleReferences[i][j].layer][const_PLLCycleReferences[i][j].cube][const_PLLCycleReferences[i][j].side];
		}
		const uint2 corner = findCorner(dev_F2LEdgeCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeIdx.x][edgeIdx.y], colors, 4);
		match[i] = corner.x == const_PLLCycleCubeReferences[i].layer && corner.y == const_PLLCycleCubeReferences[i].cube;
	}
	int j = 0;
	while (j < 4 && match[j])
	{
		j++;
	}
	return j == 4;
}

__device__ void PLLCycleSolve(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const uint2 edgeIdx)
{
	bool match[4]{};
	int idx = 0;
	while (!setPLLCycleMatch(cubeIdx, crossIdx, cornerIdx, edgeIdx, match))
	{
		const Notation* sequence = findPLLCycleSequence(match);
		executePLLCycleSequence(cubeIdx, crossIdx, cornerIdx, edgeIdx, sequence, idx);
		idx++;
	}
}

__global__ void solvePLLCycleThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x % 4;
	const int cornerIdxY = threadIdx.x / 4;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	const int edgeIdxX = threadIdx.y;
	const int edgeIdxY = threadIdx.z;
	const uint2 edgeIdx = make_uint2(edgeIdxX, edgeIdxY);

	PLLCycleSolve(cubeIdx, crossIdx, cornerIdx, edgeIdx);
}

void solvePLLCycle()
{
	solvePLLCycleThread CUDA_KERNEL(dim3(6, 4, 6), dim3(24, 4, 6))();
	hipDeviceSynchronize();
}