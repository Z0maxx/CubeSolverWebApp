#include "hip/hip_runtime.h"
#include "cross_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ int dev_crossRotations[6][4][6];

__device__ void solveCrossPiece(const int cubeIdx, const uint2 crossIdx, const int idx, const int solveOrderIdx)
{
	SolveRotation solveRotation = findSolveRotation(dev_crossRotations[cubeIdx][crossIdx.x][crossIdx.y], solveOrderIdx);
	if (solveRotation.rotation != 99)
	{
		turnCrossCube(cubeIdx, crossIdx, solveRotation.direction, solveRotation.twice);
		dev_crossRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][idx] = solveRotation.notation;
		dev_crossRotations[cubeIdx][crossIdx.x][crossIdx.y] = solveRotation.newRotation;
	}
	Color colors[2]{};
	for (int i = 0; i < 2; i++)
	{
		colors[i] = dev_crossCubeColors[cubeIdx][crossIdx.x][crossIdx.y][const_crossReferences[i].layer][const_crossReferences[i].cube][const_crossReferences[i].side];
	}
	const uint2 edge = findEdge(dev_crossCubeColors[cubeIdx][crossIdx.x][crossIdx.y], colors, 12);
	const Color targetColor = dev_crossCubeColors[cubeIdx][crossIdx.x][crossIdx.y][const_crossTargetReference.layer][const_crossTargetReference.cube][const_crossTargetReference.side];
	const Notation* sequence = findCrossSequence(edge, dev_crossCubeColors[cubeIdx][crossIdx.x][crossIdx.y][edge.x][edge.y], targetColor);
	executeCrossSequence(cubeIdx, crossIdx, sequence, idx);
}

__global__ void solveCrossThread()
{
	const int cubeIdx = blockIdx.x;
	const int crossIdxX = threadIdx.x;
	const int crossIdxY = threadIdx.y;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	for (int i = 0; i < 4; i++)
	{
		solveCrossPiece(cubeIdx, crossIdx, i, solveOrders[crossIdx.x][crossIdx.y][i]);
	}
}

__global__ void copyCrossColorThread()
{
	memcpy(dev_crossCubeColors[threadIdx.x][threadIdx.y][threadIdx.z], dev_cubeColors[threadIdx.x], sizeof(dev_cubeColors[threadIdx.x]));
}

void solveCross()
{
	copyCrossColorThread CUDA_KERNEL(1, dim3(6, 4, 6))();
	hipDeviceSynchronize();
	solveCrossThread CUDA_KERNEL(6, dim3(4, 6))();
	hipDeviceSynchronize();
}