#include "hip/hip_runtime.h"
#include "variables.cuh"
#include "f2l_corner_rotations.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__shared__ Color shr_tempF2LCornerColors[4][6][3][9][6];

__device__ void rotateF2LCornerLayerColors(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const CubeLayer cubeLayer, const Direction direction)
{
    memcpy(shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y], dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], sizeof(shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y]));

    const int alwaysMoveIdx = cubeLayer / 3;
    const int directionIdx = direction % 2;

    for (int x = 0; x < 4; x++)
    {
        const Move cornerMove = const_layerCornerMoves[cubeLayer][directionIdx][x];

        for (int y = 0; y < 2; y++)
        {
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][cornerMove.originLayer][cornerMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 4; x++)
    {
        const Move edgeMove = const_layerEdgeMoves[cubeLayer][directionIdx][x];
        shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        for (int y = 0; y < 2; y++)
        {
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_layerAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeMove.originLayer][edgeMove.originCube][const_layerAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    memcpy(dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y], sizeof(shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y]));
}

__device__ void rotateF2LCornerFaceColors(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const MoveIdx moveIdx)
{
    if (moveIdx == MoveIdx_None) return;

    const int alwaysMoveIdx = moveIdx / 2;

    for (int x = 0; x < 8; x++)
    {
        const Move cornerMove = const_faceCornerMoves[moveIdx][x];
        for (int y = 0; y < 2; y++)
        {
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][cornerMove.targetLayer][cornerMove.targetCube][cornerMove.targetSides[y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][cornerMove.originLayer][cornerMove.originCube][cornerMove.originSides[y]];
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][cornerMove.targetLayer][cornerMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][cornerMove.originLayer][cornerMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][y]];
        }
    }

    for (int x = 0; x < 12; x++)
    {
        const Move edgeMove = const_faceEdgeMoves[moveIdx][x];
        if (x < 8)
        {
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[0]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[0]];
        }
        else
        {
            for (int y = 0; y < 2; y++)
            {
                shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][edgeMove.targetLayer][edgeMove.targetCube][edgeMove.targetSides[y]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeMove.originLayer][edgeMove.originCube][edgeMove.originSides[y]];
            }
        }

        for (int i = 0; i < 2; i++)
        {
            shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][edgeMove.targetLayer][edgeMove.targetCube][const_faceAlwaysMoves[alwaysMoveIdx][i]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][edgeMove.originLayer][edgeMove.originCube][const_faceAlwaysMoves[alwaysMoveIdx][i]];
        }
    }

    for (int x = 0; x < 6; x++)
    {
        const Move centerMove = const_faceCenterMoves[moveIdx][x];
        shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y][centerMove.targetLayer][centerMove.targetCube][centerMove.targetSides[0]] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][centerMove.originLayer][centerMove.originCube][centerMove.originSides[0]];
    }

    memcpy(dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y], sizeof(shr_tempF2LCornerColors[cornerIdx.x][cornerIdx.y]));
}

__device__ void turnF2LCornerLayer(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const CubeLayer cubeLayer, const Direction direction, const bool twice) {
    rotateF2LCornerLayerColors(cubeIdx, crossIdx, cornerIdx, cubeLayer, direction);
    if (twice)
    {
        rotateF2LCornerLayerColors(cubeIdx, crossIdx, cornerIdx, cubeLayer, direction);
    }
}

__device__ void turnF2LCornerCube(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const Direction direction, const bool twice) {
    MoveIdx moveIdx = const_cubeTurnMoveIdxs[direction];
    rotateF2LCornerFaceColors(cubeIdx, crossIdx, cornerIdx, moveIdx);
    if (twice)
    {
        rotateF2LCornerFaceColors(cubeIdx, crossIdx, cornerIdx, moveIdx);
    }
}