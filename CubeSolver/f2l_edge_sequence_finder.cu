#include "hip/hip_runtime.h"
#include "f2l_edge_sequence_finder.cuh"

__device__ const Notation* findF2LEdgeSequence(const uint2 edge, const Color colors[6], const Color targetColor)
{
	for (int i = 0; i < 16; i++)
	{
		if (const_F2LEdgeSequences[i].layer == edge.x && const_F2LEdgeSequences[i].cube == edge.y && colors[const_F2LEdgeSequences[i].side] == targetColor)
		{
			return const_F2LEdgeSequences[i].sequence;
		}
	}
	return { 0 };
}