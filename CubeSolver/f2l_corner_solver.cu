#include "hip/hip_runtime.h"
#include "f2l_corner_solver.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ int dev_F2LCornerRotations[6][4][6][4][6];

__device__ void solveF2LCornerPiece(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const int idx, const int solveOrderIdx)
{
	SolveRotation solveRotation = findSolveRotation(dev_F2LCornerRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], solveOrderIdx);
	if (solveRotation.rotation != 99)
	{
		turnF2LCornerCube(cubeIdx, crossIdx, cornerIdx, solveRotation.direction, solveRotation.twice);
		dev_F2LCornerRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][idx] = solveRotation.notation;
		dev_F2LCornerRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y] = solveRotation.newRotation;
	}
	Color colors[3]{};
	for (int i = 0; i < 3; i++)
	{
		colors[i] = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][const_F2LCornerReferences[i].layer][const_F2LCornerReferences[i].cube][const_F2LCornerReferences[i].side];
	}
	const uint2 corner = findCorner(dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], colors, 8);
	const Color targetColor = dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][const_F2LCornerTargetReference.layer][const_F2LCornerTargetReference.cube][const_F2LCornerTargetReference.side];
	const Notation* sequence = findCornerSequence(const_F2LCornerSequences, corner, dev_F2LCornerCubeColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][corner.x][corner.y], targetColor);
	executeF2LCornerSequence(cubeIdx, crossIdx, cornerIdx, sequence, idx);
}

__global__ void solveF2LCornerThread()
{
	const int cubeIdx = blockIdx.x;

	const int crossIdxX = blockIdx.y;
	const int crossIdxY = blockIdx.z;
	const uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	const int cornerIdxX = threadIdx.x;
	const int cornerIdxY = threadIdx.y;
	const uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	for (int i = 0; i < 4; i++)
	{
		solveF2LCornerPiece(cubeIdx, crossIdx, cornerIdx, i, solveOrders[cornerIdx.x][cornerIdx.y][i]);
	}
}

__global__ void copyF2LCornerColorThread()
{
	memcpy(dev_F2LCornerCubeColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y], dev_crossCubeColors[blockIdx.x][blockIdx.y][blockIdx.z], sizeof(dev_crossCubeColors[blockIdx.x][blockIdx.y][blockIdx.z]));
}

void solveF2LCorner()
{
	copyF2LCornerColorThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6))();
	hipDeviceSynchronize();
	solveF2LCornerThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6))();
	hipDeviceSynchronize();
}