#include "hip/hip_runtime.h"
#include "f2l_corner_solver.cuh"

__device__ int dev_F2LCornerRotations[6][4][6][4][6];

__device__ void solveF2LCornerPiece(const int cubeIdx, const uint2 crossIdx, const uint2 cornerIdx, const int idx, const int solveOrderIdx)
{
	SolveRotation solveRotation = findSolveRotation(dev_F2LCornerRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], solveOrderIdx);
	if (solveRotation.rotation != 99)
	{
		turnF2LCornerCube(cubeIdx, crossIdx, cornerIdx, solveRotation.direction, solveRotation.twice);
		dev_F2LCornerRotationMoves[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][idx] = solveRotation.notation;
		dev_F2LCornerRotations[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y] = solveRotation.newRotation;
	}
	Color colors[3]{};
	for (int i = 0; i < 3; i++)
	{
		colors[i] = dev_F2LCornerColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][const_F2LCornerReferences[i].layer][const_F2LCornerReferences[i].cube][const_F2LCornerReferences[i].side];
	}
	uint2 corner = findCorner(dev_F2LCornerColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y], colors, 8);
	Color targetColor = dev_F2LCornerColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][const_F2LCornerTargetReference.layer][const_F2LCornerTargetReference.cube][const_F2LCornerTargetReference.side];
	const Notation* sequence = findCornerSequence(const_F2LCornerSequences, corner, dev_F2LCornerColors[cubeIdx][crossIdx.x][crossIdx.y][cornerIdx.x][cornerIdx.y][corner.x][corner.y], targetColor);
	executeF2LCornerSequence(cubeIdx, crossIdx, cornerIdx, sequence, idx);
}

__global__ void solveF2LCornerThread()
{
	int cubeIdx = blockIdx.x;

	int crossIdxX = blockIdx.y;
	int crossIdxY = blockIdx.z;
	uint2 crossIdx = make_uint2(crossIdxX, crossIdxY);

	int cornerIdxX = threadIdx.x;
	int cornerIdxY = threadIdx.y;
	uint2 cornerIdx = make_uint2(cornerIdxX, cornerIdxY);

	for (int i = 0; i < 4; i++)
	{
		solveF2LCornerPiece(cubeIdx, crossIdx, cornerIdx, i, const_solveOrders[cornerIdx.x][cornerIdx.y][i]);
	}
}

__global__ void copyF2LCornerColorThread()
{
	memcpy(dev_F2LCornerColors[blockIdx.x][blockIdx.y][blockIdx.z][threadIdx.x][threadIdx.y], dev_crossColors[blockIdx.x][blockIdx.y][blockIdx.z], sizeof(dev_crossColors[blockIdx.x][blockIdx.y][blockIdx.z]));
}

void solveF2LCorner()
{
	copyF2LCornerColorThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6))();
	hipDeviceSynchronize();
	solveF2LCornerThread CUDA_KERNEL(dim3(6, 4, 6), dim3(4, 6))();
	hipDeviceSynchronize();
}