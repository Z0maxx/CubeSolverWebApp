#include "hip/hip_runtime.h"
#include "rgb_cube_color_converter.cuh"

__device__ CubeColor convertRgbToCubeColor(int r, int g, int b)
{
	float R = r;
	float G = g;
	float B = b;
	float sumVal = (R + G + B);
	float I = (float)sumVal / 3;
	float minval = fmin(fmin(R, G), B);
	float S = 1 - 3 * minval / sumVal;
	float top = ((R - G) + (R - B)) / 2;
	float bottom = sqrt(pow(R - G, 2) + (R - B) * (G - B)) + 0.0000001;
	float theta = acos(top / bottom);

	float H = theta;
	if (B > G)
	{
		H = 360 * PI / 180 - theta;
	}
	H = H * 180 / PI;

	float deg = H;
	if (I > 150 && S <= 0.25)
	{
		return CubeColor_White;
	}
	if (I > 20 && S > 0.25)
	{
		if (360 - deg < 10 || deg <= 10)
		{
			return CubeColor_Red;
		}
		if (deg > 10 && deg <= 40)
		{
			return CubeColor_Orange;
		}
		if (deg > 40 && deg <= 70)
		{
			return CubeColor_Yellow;
		}
		if (deg > 70 && deg <= 160)
		{
			return CubeColor_Green;
		}
		if (deg > 160 && deg <= 260)
		{
			return CubeColor_Blue;
		}
	}
	return CubeColor_None;
}