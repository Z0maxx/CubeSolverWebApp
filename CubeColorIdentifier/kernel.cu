﻿
#include "cube_color_identifier.cuh"

int main(int argc, char** argv)
{
	hipSetDeviceFlags(hipDeviceScheduleSpin);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	threadsPerBlock = prop.maxThreadsPerBlock;
	int baseWidth = atoi(argv[1]);
	int baseHeight = atoi(argv[2]);
	setVariables(baseWidth, baseHeight);
	fileName = argv[3];
	char* imageChar = (char*)malloc(baseImageLength * sizeof(char));
	FILE* fileptr = fopen(fileName, "rb");
	fread(imageChar, sizeof(char), baseImageLength, fileptr);
	int* image = (int*)malloc(baseImageLength * sizeof(int));
	for (int i = 0; i < baseImageLength; i++)
	{
		image[i] = imageChar[i] < 0 ? imageChar[i] + 256 : imageChar[i];
	}
	fclose(fileptr);
	hipEvent_t start, stop;
	float t;
	identifyColors(image);
	return 0;
}