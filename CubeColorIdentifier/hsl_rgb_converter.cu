#include "hip/hip_runtime.h"
#include "hsl_rgb_converter.cuh"

float hueToRgb(float p, float q, float t) {
	if (t < 0) t += 1;
	if (t > 1) t -= 1;
	if (t < 1.0 / 6) return p + (q - p) * 6 * t;
	if (t < 1.0 / 2) return q;
	if (t < 2.0 / 3) return p + (q - p) * (2.0 / 3 - t) * 6;
	return p;
}

dim3 convertHslToRgb(float h, float s, float l) {
	float r = 0;
	float g = 0;
	float b = 0;

	if (s == 0) {
		r = g = b = l;
	}
	else {
		float q = l < 0.5 ? l * (1 + s) : l + s - l * s;
		float p = 2 * l - q;
		r = hueToRgb(p, q, h + 1.0 / 3);
		g = hueToRgb(p, q, h);
		b = hueToRgb(p, q, h - 1.0 / 3);
	}

	return dim3(round(r * 255), round(g * 255), round(b * 255));
}