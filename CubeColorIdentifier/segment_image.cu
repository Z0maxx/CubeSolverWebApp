#include "hip/hip_runtime.h"
#include "segment_image.cuh"

__global__ void segmentPixel(int** dev_image, int** dev_segmentedImage)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);

	bool diff = false;

	int i = y - 1;
	while (!diff && i <= y + 1)
	{
		int j = x - 1;
		while (!diff && j <= x + 1)
		{
			if (i >= 0 && i < const_height && j >= 0 && j < const_width)
			{
				if (abs(dev_image[y][x] - dev_image[i][j]) > 2)
				{
					dev_segmentedImage[y][x] = 255;
					diff = true;
				}
			}
			j++;
		}
		i++;
	}
}

int** segmentImage(int** dev_image)
{
	int** dev_segmentedImage;
	hipMalloc(&dev_segmentedImage, heightSize);
	int** segmentedImage = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&segmentedImage[i], width * sizeof(int));
	}
	hipMemcpy(dev_segmentedImage, segmentedImage, heightSize, hipMemcpyHostToDevice);

	segmentPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_segmentedImage);
	hipDeviceSynchronize();
	
	return dev_segmentedImage;
}