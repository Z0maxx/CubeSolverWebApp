#include "hip/hip_runtime.h"
#include "min_max_coords.cuh"

__global__ void minMaxCoordPixel(int** dev_image, int* dev_minX, int* dev_minY, int* dev_maxX, int* dev_maxY)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength || y >= const_height - 1 || y <= 0 || x >= const_width - 1 || x <= 0) return;

	if (dev_image[y][x] == 0 && (dev_image[y + 1][x] == 255 || dev_image[y - 1][x] == 255 || dev_image[y][x - 1] == 255 || dev_image[y][x + 1] == 255))
	{
		atomicMin(dev_minX, x);
		atomicMin(dev_minY, y);
		atomicMax(dev_maxX, x);
		atomicMax(dev_maxY, y);
	}
}

void minMaxCoords(int** dev_image, int* minX, int* minY, int* maxX, int* maxY)
{
	*minX = width - 1;
	*minY = height - 1;
	int* dev_minX;
	int* dev_minY;
	int* dev_maxX;
	int* dev_maxY;
	hipMalloc(&dev_minX, sizeof(int*));
	hipMalloc(&dev_minY, sizeof(int*));
	hipMalloc(&dev_maxX, sizeof(int*));
	hipMalloc(&dev_maxY, sizeof(int*));
	hipMemcpy(dev_minX, minX, sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(dev_minY, minY, sizeof(int*), hipMemcpyHostToDevice);
	minMaxCoordPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_minX, dev_minY, dev_maxX, dev_maxY);
	hipDeviceSynchronize();
	hipMemcpy(minX, dev_minX, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(minY, dev_minY, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(maxX, dev_maxX, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(maxY, dev_maxY, sizeof(int*), hipMemcpyDeviceToHost);
	*minX -= 8;
	*minY -= 8;
	*maxX += 8;
	*maxY += 8;
	if (*minX < 0) *minX = 0;
	if (*minY < 0) *minY = 0;
	if (*maxX >= width) *maxX = width - 1;
	if (*maxY >= height) *maxY = height - 1;
}