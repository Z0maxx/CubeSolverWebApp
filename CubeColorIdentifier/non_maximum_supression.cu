#include "hip/hip_runtime.h"
#include "non_maximum_supression.cuh"

__global__ void nonMaximumSupressionPixel(float** dev_angles, int** dev_gradient, int** dev_supressed)
{

	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	float angle = dev_angles[y][x];
	if (angle < 0) angle += 180;

	if (x > 0 && x < const_width - 1 && y > 0 && y < const_height - 1)
	{
		int a = 0;
		int b = 0;
		if (angle < 22.5 || angle >= 157.5)
		{
			a = dev_gradient[y][x - 1];
			b = dev_gradient[y][x + 1];
		}
		else if (angle >= 22.5 && angle < 67.5)
		{
			a = dev_gradient[y + 1][x + 1];
			b = dev_gradient[y - 1][x - 1];
		}
		else if (angle >= 67.5 && angle < 112.5)
		{
			a = dev_gradient[y + 1][x];
			b = dev_gradient[y - 1][x];
		}
		else if (angle >= 112.5 && angle < 157.5)
		{
			a = dev_gradient[y + 1][x - 1];
			b = dev_gradient[y - 1][x + 1];
		}

		int current = dev_gradient[y][x];
		if (current > a && current > b)
		{
			dev_supressed[y][x] = current;
		}
		else
		{
			dev_supressed[y][x] = 0;
		}
	}
	else
	{
		dev_supressed[y][x] = dev_gradient[y][x];
	}
}

int** nonMaximumSupression(int** dev_gradientX, int** dev_gradientY, int** dev_gradient, float** dev_angles)
{
	int** dev_supressed;
	hipMalloc(&dev_supressed, heightSize);
	int** supressed = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&supressed[i], widthIntSize);
	}
	hipMemcpy(dev_supressed, supressed, heightSize, hipMemcpyHostToDevice);

	nonMaximumSupressionPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_angles, dev_gradient, dev_supressed);
	hipDeviceSynchronize();

	return dev_supressed;
}