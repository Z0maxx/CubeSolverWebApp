#include "hip/hip_runtime.h"
#include "reduce_mean_shifted.cuh"

__global__ void reduceMeanShiftedPixel(int** dev_blurred, int** dev_reducedBlurred, int** dev_meanShifted, int** dev_reducedMeanShifted, int minX, int minY)
{
	int idx = getIdx();
	if (idx >= const_imageLength) return;

	int y1 = getY(idx);
	int y2 = y1 + minY;
	int x1 = getX(idx);
	int x2 = x1 * 3;
	int x3 = (x1 + minX) * 3;

	dev_reducedBlurred[y1][x1] = dev_blurred[y2][x1 + minX];
	dev_reducedMeanShifted[y1][x2] = dev_meanShifted[y2][x3];
	dev_reducedMeanShifted[y1][x2 + 1] = dev_meanShifted[y2][x3 + 1];
	dev_reducedMeanShifted[y1][x2 + 2] = dev_meanShifted[y2][x3 + 2];
}

void reduceMeanShifted(int** dev_blurred, int** dev_reducedBlurred, int** dev_meanShifted, int** dev_reducedMeanShifted, int minX, int minY)
{
	int** reducedBlurred = (int**)malloc(heightSize);
	int** reducedMeanShifted = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&reducedBlurred[i], widthIntSize);
		hipMalloc(&reducedMeanShifted[i], widthRGBSize);
	}
	hipMemcpy(dev_reducedBlurred, reducedBlurred, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_reducedMeanShifted, reducedMeanShifted, heightSize, hipMemcpyHostToDevice);

	reduceMeanShiftedPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_blurred, dev_reducedBlurred, dev_meanShifted, dev_reducedMeanShifted, minX, minY);
	hipDeviceSynchronize();
}