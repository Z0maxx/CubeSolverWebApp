#include "hip/hip_runtime.h"
#include "hough_image.cuh"

__global__ void houghImageLine(int** dev_image, HoughLine* dev_houghLines, bool* dev_accepted)
{
	if (!dev_accepted[threadIdx.x]) return;

	HoughLine line = dev_houghLines[threadIdx.x];
	int rho = line.rho;
	float radian = line.theta * PI / 180;
	float sinRes = sin(radian);
	float cosRes = cos(radian);

	for (int i = 0; i < const_width; i++)
	{
		int ycoord = round((rho - i * cosRes) / sinRes);
		if (ycoord >= 0 && ycoord < const_height)
		{
			dev_image[ycoord][i] = 255;
		}
	}

	for (int i = 0; i < const_height; i++)
	{
		int xcoord = round((rho - i * sinRes) / cosRes);
		if (xcoord >= 0 && xcoord < const_width)
		{
			dev_image[i][xcoord] = 255;
		}
	}
}

bool compHough(HoughLine line1, HoughLine line2)
{
	if (line1.rho == line2.rho)
	{
		return line1.theta < line2.theta;
	}
	return line1.rho < line2.rho;
}

std::vector<HoughLine> groupLines(std::vector<HoughLine> houghLines, float range, float angle)
{
	std::vector<HoughLine> uniqueLines;
	std::vector<std::vector<HoughLine>> lineGroups;

	for (int i = 0; i < houghLines.size(); i++)
	{
		HoughLine line = houghLines[i];
		bool unique = true;
		int rho = line.rho;
		int theta = line.theta;

		int j = 0;
		while (unique && j < uniqueLines.size())
		{
			int uniqueRho = uniqueLines[j].rho;
			int uniqueTheta = uniqueLines[j].theta;
			if (abs(uniqueRho - rho) < range && abs(uniqueTheta - theta) < angle)
			{
				unique = false;
			}
			else
			{
				j++;
			}
		}

		if (unique)
		{
			uniqueLines.push_back(line);
			std::vector<HoughLine> group;
			group.push_back(line);
			lineGroups.push_back(group);
		}
		else
		{
			lineGroups[j].push_back(line);
		}
	}

	for (int i = 0; i < uniqueLines.size(); i++)
	{
		int sumRho = 0;
		int sumTheta = 0;

		std::vector<HoughLine> lineGroup = lineGroups[i];
		int groupLength = lineGroup.size();

		for (int j = 0; j < groupLength; j++)
		{
			HoughLine line = lineGroup[j];
			sumRho += line.rho;
			sumTheta += line.theta;
		}

		uniqueLines[i] = { (int)round((float)sumRho / groupLength), (int)round((float)sumTheta / groupLength) };
	}

	return uniqueLines;
}

__device__ int getRhoDiff(HoughLine line1, HoughLine line2)
{
	return abs(line1.rho - line2.rho);
}

__device__ int getThetaDiff(HoughLine line1, HoughLine line2)
{
	return abs(line1.theta - line2.theta);
}

__global__ void setAcceptedLines(HoughLine* dev_lines, bool* dev_accepted, int length)
{
	int lineIdx = threadIdx.x;
	HoughLine line1 = dev_lines[lineIdx];
	bool accepted = false;
	int i = lineIdx + 1;
	while (!accepted && i < length)
	{
		HoughLine line2 = dev_lines[i];
		int rhoDiff1 = getRhoDiff(line1, line2);
		int thetaDiff1 = getThetaDiff(line1, line2);
		if (thetaDiff1 < 4)
		{
			int j = i + 1;
			while (!accepted && j < length)
			{
				HoughLine line3 = dev_lines[j];
				int rhoDiff2 = getRhoDiff(line2, line3);
				int thetaDiff2 = getThetaDiff(line2, line3);
				if (abs(rhoDiff1 - rhoDiff2) < rhoDiff1 * 0.18 && thetaDiff2 < 4)
				{
					int k = j + 1;
					while (!accepted && k < length)
					{
						HoughLine line4 = dev_lines[k];
						int rhoDiff3 = getRhoDiff(line3, line4);
						int thetaDiff3 = getThetaDiff(line3, line4);
						if (abs(rhoDiff2 - rhoDiff3) < rhoDiff2 * 0.18 && thetaDiff3 < 4)
						{
							accepted = true;
							dev_accepted[lineIdx] = true;
							dev_accepted[i] = true;
							dev_accepted[j] = true;
							dev_accepted[k] = true;
						}
						k++;
					}
				}
				j++;
			}
		}
		i++;
	}
}

int** houghImage(std::vector<HoughLine> houghLines, int** image, int** thresholded, int** dev_thresholded)
{
	std::sort(houghLines.begin(), houghLines.end(), compHough);
	for (int i = 0; i < houghLines.size(); i++)
	{
		int rho = houghLines[i].rho;
		if (rho < 0)
		{
			houghLines[i].rho = -rho;
			houghLines[i].theta += 180;
		}
	}

	int** dev_image;
	hipMalloc(&dev_image, heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&image[i], widthIntSize);
	}
	hipMemcpy(dev_image, image, heightSize, hipMemcpyHostToDevice);

	int maxCount = 0;
	HoughLine* dev_lines1;
	HoughLine* dev_lines2;
	bool* dev_accepted1;
	bool* dev_accepted2;
	int length1 = 0;
	int length2 = 0;
	for (int tryIdx = 0; tryIdx < 2; tryIdx++)
	{
		std::vector<HoughLine> uniqueLines = groupLines(houghLines, tryIdx == 0 ? 22 : 20, tryIdx == 0 ? 8 : 10);

		int length = uniqueLines.size();
		int size = length * sizeof(HoughLine);
		HoughLine* dev_lines;
		hipMalloc(&dev_lines, size);
		HoughLine* lines = (HoughLine*)malloc(size);
		for (int i = 0; i < length; i++)
		{
			lines[i] = uniqueLines[i];
		}
		hipMemcpy(dev_lines, lines, size, hipMemcpyHostToDevice);


		bool* dev_accepted;
		hipMalloc(&dev_accepted, length * sizeof(bool));
		setAcceptedLines CUDA_KERNEL(1, length)(dev_lines, dev_accepted, length);
		hipDeviceSynchronize();
		if (tryIdx == 0)
		{
			length1 = length;
			dev_accepted1 = dev_accepted;
			dev_lines1 = dev_lines;
		}
		else
		{
			length2 = length;
			dev_accepted2 = dev_accepted;
			dev_lines2 = dev_lines;
		}

		bool* accepted = (bool*)malloc(length * sizeof(bool));
		hipMemcpy(accepted, dev_accepted, length * sizeof(bool), hipMemcpyDeviceToHost);
		int count = 0;
		for (int i = 0; i < length; i++)
		{
			if (accepted[i] == true) count++;
		}
		if (count > maxCount)
		{
			maxCount = count;
		}
		if (count >= 8)
		{
			houghImageLine CUDA_KERNEL(1, length)(dev_image, dev_lines, dev_accepted);
			hipDeviceSynchronize();

			tryIdx = 2;
		}
	}

	if (maxCount < 8)
	{
		houghImageLine CUDA_KERNEL(1, length1)(dev_image, dev_lines1, dev_accepted1);
		hipDeviceSynchronize();

		houghImageLine CUDA_KERNEL(1, length2)(dev_image, dev_lines2, dev_accepted2);
		hipDeviceSynchronize();
	}

	return dev_image;
}