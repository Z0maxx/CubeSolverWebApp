#include "hip/hip_runtime.h"
#include "hough_image.cuh"

__global__ void houghImageLine(int** dev_image, HoughLine* dev_houghLines, bool* dev_accepted)
{
	if (!dev_accepted[threadIdx.x]) return;

	HoughLine line = dev_houghLines[threadIdx.x];
	int rho = line.rho;
	float radian = line.theta * PI / 180;
	float sinRes = sin(radian);
	float cosRes = cos(radian);

	for (int i = 0; i < const_width; i++)
	{
		int ycoord = round((rho - i * cosRes) / sinRes);
		if (ycoord >= 0 && ycoord < const_height)
		{
			dev_image[ycoord][i] = 255;
		}
	}

	for (int i = 0; i < const_height; i++)
	{
		int xcoord = round((rho - i * sinRes) / cosRes);
		if (xcoord >= 0 && xcoord < const_width)
		{
			dev_image[i][xcoord] = 255;
		}
	}
}

bool compHough(HoughLine line1, HoughLine line2)
{
	if (line1.rho == line2.rho)
	{
		return line1.theta < line2.theta;
	}
	return line1.rho < line2.rho;
}

std::vector<HoughLine> groupLines(std::vector<HoughLine> houghLines, float range, float angle)
{
	std::vector<HoughLine> uniqueLines;
	std::vector<std::vector<HoughLine>> lineGroups;

	for (int i = 0; i < houghLines.size(); i++)
	{
		HoughLine line = houghLines[i];
		bool unique = true;
		int rho = line.rho;
		int theta = line.theta;

		int j = 0;
		while (unique && j < uniqueLines.size())
		{
			int uniqueRho = uniqueLines[j].rho;
			int uniqueTheta = uniqueLines[j].theta;
			if (abs(uniqueRho - rho) < range && abs(uniqueTheta - theta) < angle)
			{
				unique = false;
			}
			else
			{
				j++;
			}
		}

		if (unique)
		{
			uniqueLines.push_back(line);
			std::vector<HoughLine> group;
			group.push_back(line);
			lineGroups.push_back(group);
		}
		else
		{
			lineGroups[j].push_back(line);
		}
	}

	for (int i = 0; i < uniqueLines.size(); i++)
	{
		int sumRho = 0;
		int sumTheta = 0;

		std::vector<HoughLine> lineGroup = lineGroups[i];
		int groupLength = lineGroup.size();

		for (int j = 0; j < groupLength; j++)
		{
			HoughLine line = lineGroup[j];
			sumRho += line.rho;
			sumTheta += line.theta;
		}

		uniqueLines[i] = { (int)round((float)sumRho / groupLength), (int)round((float)sumTheta / groupLength) };
	}

	return uniqueLines;
}

__device__ int getRhoDiff(HoughLine line1, HoughLine line2)
{
	return abs(line1.rho - line2.rho);
}

__device__ int getThetaDiff(HoughLine line1, HoughLine line2)
{
	return abs(line1.theta - line2.theta);
}

__global__ void setAcceptedLines(HoughLine* dev_lines, bool* dev_accepted, int length)
{
	int lineIdx = threadIdx.x;
	HoughLine line1 = dev_lines[lineIdx];
	bool accepted = false;
	int i = lineIdx + 1;
	while (!accepted && i < length)
	{
		HoughLine line2 = dev_lines[i];
		int rhoDiff1 = getRhoDiff(line1, line2);
		int thetaDiff1 = getThetaDiff(line1, line2);
		if (thetaDiff1 < 8)
		{
			int j = i + 1;
			while (!accepted && j < length)
			{
				HoughLine line3 = dev_lines[j];
				int rhoDiff2 = getRhoDiff(line2, line3);
				int thetaDiff2 = getThetaDiff(line2, line3);
				if (abs(rhoDiff1 - rhoDiff2) < rhoDiff1 * 0.15 && thetaDiff2 < 5)
				{
					int k = j + 1;
					while (!accepted && k < length)
					{
						HoughLine line4 = dev_lines[k];
						int rhoDiff3 = getRhoDiff(line3, line4);
						int thetaDiff3 = getThetaDiff(line3, line4);
						if (abs(rhoDiff2 - rhoDiff3) < rhoDiff2 * 0.15 && thetaDiff3 < 5)
						{
							accepted = true;
							dev_accepted[lineIdx] = true;
							dev_accepted[i] = true;
							dev_accepted[j] = true;
							dev_accepted[k] = true;
						}
						k++;
					}
				}
				j++;
			}
		}
		i++;
	}
}

int** houghImage(std::vector<HoughLine> houghLines, int** image, int** thresholded, int** dev_thresholded)
{
	for (int i = 0; i < houghLines.size(); i++)
	{
		int rho = houghLines[i].rho;
		if (rho < 0)
		{
			houghLines[i].rho = -rho;
			houghLines[i].theta += 180;
		}
	}
	std::sort(houghLines.begin(), houghLines.end(), compHough);
	std::vector<HoughLine> uniqueLines = groupLines(houghLines, 20, 10);

	int length = uniqueLines.size();
	int size = length * sizeof(HoughLine);
	HoughLine* dev_lines;
	hipMalloc(&dev_lines, size);
	HoughLine* lines = (HoughLine*)malloc(size);
	for (int i = 0; i < length; i++)
	{
		lines[i] = uniqueLines[i];
	}
	hipMemcpy(dev_lines, lines, size, hipMemcpyHostToDevice);

	bool* dev_accepted;
	hipMalloc(&dev_accepted, length * sizeof(bool));
	setAcceptedLines CUDA_KERNEL(1, length - 3)(dev_lines, dev_accepted, length);
	hipDeviceSynchronize();

	HoughLine* dev_houghLines;
	hipMalloc(&dev_houghLines, size);
	HoughLine* uniqueHoughLines = (HoughLine*)malloc(size);
	for (int i = 0; i < length; i++)
	{
		uniqueHoughLines[i] = uniqueLines[i];
	}
	hipMemcpy(dev_houghLines, uniqueHoughLines, size, hipMemcpyHostToDevice);

	int** dev_image;
	hipMalloc(&dev_image, heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&image[i], widthIntSize);
	}
	hipMemcpy(dev_image, image, heightSize, hipMemcpyHostToDevice);

	houghImageLine CUDA_KERNEL(1, length)(dev_image, dev_houghLines, dev_accepted);
	hipDeviceSynchronize();

	return dev_image;
}