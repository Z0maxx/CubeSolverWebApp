#include "hip/hip_runtime.h"
#include "hysteresis_threshold.cuh"

__global__ void hysteresisThresholdPixel(int** dev_image, int** dev_thresholded, int lowerLimit, int upperLimit)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	if (x > 0 && x < const_width - 1 && y > 0 && y < const_height - 1)
	{
		if (dev_image[y][x] < lowerLimit)
		{
			dev_thresholded[y][x] = 0;
		}
		else
		{
			bool needed = false;
			int i = y - 1;
			int j = x - 1;
			int checked = 0;
			while (!needed && checked < 9)
			{
				if (dev_image[i][j] > upperLimit)
				{
					needed = true;
				}
				checked++;
				j++;
				if (j > x + 1)
				{
					i++;
					j = x - 1;
				}
			}
			if (needed)
			{
				dev_thresholded[y][x] = 255;
			}
			else
			{
				dev_thresholded[y][x] = 0;
			}
		}
	}
	else
	{
		dev_thresholded[y][x] = 0;
	}
}

int** hysteresisThreshold(int** dev_image, int** thresholded, int lowerLimit, int upperLimit)
{
	int** dev_thresholded;
	hipMalloc(&dev_thresholded, heightSize);
	
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&thresholded[i], widthIntSize);
	}
	hipMemcpy(dev_thresholded, thresholded, heightSize, hipMemcpyHostToDevice);

	hysteresisThresholdPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_thresholded, lowerLimit, upperLimit);
	hipDeviceSynchronize();

	return dev_thresholded;
}