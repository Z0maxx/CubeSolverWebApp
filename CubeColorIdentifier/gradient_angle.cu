#include "hip/hip_runtime.h"
#include "gradient_angle.cuh"

__global__ void gradientAnglePixel(int** dev_sobelX, int** dev_sobelY, float** dev_angles)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	float angle = 180.0 / PI * atan2((float)dev_sobelY[y][x], (float)((float)dev_sobelX[y][x] + 0.0000000001));
	dev_angles[y][x] = angle;
}

float** gradientAngle(int** dev_gradientX, int** dev_gradientY)
{
	float** dev_angles;
	hipMalloc(&dev_angles, heightSize);
	float** angles = (float**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&angles[i], widthDoubleSize);
	}
	hipMemcpy(dev_angles, angles, heightSize, hipMemcpyHostToDevice);
	
	gradientAnglePixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_gradientX, dev_gradientY, dev_angles);
	hipDeviceSynchronize();

	return dev_angles;
}