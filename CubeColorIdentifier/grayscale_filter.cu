#include "hip/hip_runtime.h"
#include "grayscale_filter.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__global__ void grayscalePixel(int** dev_image, int** dev_grayscaled)
{
	int idx = getIdx();
	if (idx >= const_imageLength) return;
	int y = getY(idx);
	int x1 = getX(idx);
	int x2 = x1 * 3;
	float blue = dev_image[y][x2 + 2];
	float green = dev_image[y][x2 + 1];
	float red = dev_image[y][x2];
	int res = (int)(0.299 * red + 0.587 * green + 0.114 * blue);
	dev_grayscaled[y][x1] = res;
}

int** grayscale(int** dev_image)
{
	int** dev_grayscaled;
	hipMalloc(&dev_grayscaled, heightSize);
	int** grayscaled = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&grayscaled[i], widthIntSize);
	}
	hipMemcpy(dev_grayscaled, grayscaled, heightSize, hipMemcpyHostToDevice);
	
	grayscalePixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_grayscaled);
	hipDeviceSynchronize();

	return dev_grayscaled;
}