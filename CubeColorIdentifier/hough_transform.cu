#include "hip/hip_runtime.h"
#include "hough_transform.cuh"

__global__ void linePixel(int** dev_image, int** dev_votes, int maxRho, int minVotes)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);

	if (idx >= const_imageLength || dev_image[y][x] == 0) return;

	int angle = blockIdx.y;
	float radian = (angle - 90) * PI / 180;
	int distance = round(y * sin(radian) + x * cos(radian));
	float cosRes = cos(radian);
	float sinRes = sin(radian);
	atomicAdd(&(dev_votes[maxRho + distance][angle]), 1);
}

std::vector<HoughLine> houghTransform(int** dev_image, float** dev_angles, int minPoints)
{
	int maxRho = ceil(sqrt(pow((float)height, 2) + pow((float)width, 2)));
	int doubleRho = 2 * maxRho;
	int doubleRhoSize = doubleRho * sizeof(int*);
	int angles = 180;
	int angleSize = angles * sizeof(int);

	int** dev_votes;
	hipMalloc(&dev_votes, doubleRhoSize);
	int** votes = (int**)malloc(doubleRhoSize);
	
	for (int i = 0; i < doubleRho; i++)
	{
		hipMalloc(&votes[i], angleSize);
	}
	hipMemcpy(dev_votes, votes, doubleRhoSize, hipMemcpyHostToDevice);

	linePixel CUDA_KERNEL(dim3(blocks, angles), threadsPerBlock)(dev_image, dev_votes, maxRho, minPoints);
	hipDeviceSynchronize();

	std::vector<HoughLine> houghLines;
	int** resVotes = (int**)malloc(doubleRhoSize);
	for (int i = 0; i < doubleRho; i++)
	{
		resVotes[i] = (int*)malloc(angleSize);
		hipMemcpy(resVotes[i], votes[i], angleSize, hipMemcpyDeviceToHost);
		for (int j = 0; j < angles; j++)
		{
			if (resVotes[i][j] >= minPoints)
			{
				houghLines.push_back({ i - maxRho, j - 90 });
			}
		}
	}

	return houghLines;
}