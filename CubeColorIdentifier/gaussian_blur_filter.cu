#include "hip/hip_runtime.h"
#include "gaussian_blur_filter.cuh"

__global__ void gaussianBlurPixel1(int** dev_image, float** dev_blurred, float* dev_kernel, int radius, int kernelLength)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	float sum = 0.0;
	int i = x - radius;
	for (int j = 0; j < kernelLength; j++)
	{
		if (i >= 0 && i < const_width)
		{
			sum += dev_image[y][i] * dev_kernel[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_image[y][-i];
			}
			else
			{
				mirror = dev_image[y][const_width - (i - const_width + 2)];
			}
			sum += mirror * dev_kernel[j];
		}
		i++;
	}
	dev_blurred[y][x] = sum;
}

__global__ void gaussianBlurPixel2(float** dev_blurred1, int** dev_blurred2, float* dev_kernel, int radius, int kernelLength)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	float sum = 0.0;
	int i = y - radius;
	for (int j = 0; j < kernelLength; j++)
	{
		if (i >= 0 && i < const_height)
		{
			sum += dev_blurred1[i][x] * dev_kernel[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_blurred1[-i][x];
			}
			else
			{
				mirror = dev_blurred1[const_height - (i - const_height + 2)][x];
			}
			sum += mirror * dev_kernel[j];
		}
		i++;
	}
	dev_blurred2[y][x] = sum;
}

float getGaussianValue(int pos, float sigma)
{
	float sigmaSquared = sigma * sigma;
	return exp(-pow(pos, 2) / (2 * sigmaSquared));
}

int getKernelLength(float sigma)
{
	int kernelLength = (int)(2 * PI * sigma);
	if (kernelLength % 2 == 0) kernelLength++;
	return kernelLength;
}

float* getKernel(float sigma)
{
	int kernelLength = getKernelLength(sigma);
	float* kernel = new float[kernelLength];
	kernel[kernelLength / 2] = getGaussianValue(0, sigma);
	int pos = 1;
	float sum = 1.0;
	while (pos <= kernelLength / 2)
	{
		float value = getGaussianValue(pos, sigma);
		kernel[kernelLength / 2 - pos] = value;
		kernel[kernelLength / 2 + pos] = value;
		sum += 2 * value;
		pos++;
	}
	float alfa = 1.0 / sum;
	for (int i = 0; i < kernelLength; i++)
	{
		kernel[i] = kernel[i] * alfa;
	}
	return kernel;
}

int** gaussianBlur(int** dev_image, float sigma)
{
	int kernelLength = getKernelLength(sigma);
	int kernelSize = kernelLength * sizeof(float);
	float* dev_kernel;
	hipMalloc(&dev_kernel, kernelSize);
	float* kernel = getKernel(sigma);
	hipMemcpy(dev_kernel, kernel, kernelSize, hipMemcpyHostToDevice);
	int radius = (kernelLength - 3) / 2 + 1;

	float** dev_blurred1;
	int** dev_blurred2;
	hipMalloc(&dev_blurred1, heightSize);
	hipMalloc(&dev_blurred2, heightSize);
	float** blurred1 = (float**)malloc(heightSize);
	int** blurred2 = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&blurred1[i], widthDoubleSize);
		hipMalloc(&blurred2[i], widthIntSize);
	}
	hipMemcpy(dev_blurred1, blurred1, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_blurred2, blurred2, heightSize, hipMemcpyHostToDevice);
	
	gaussianBlurPixel1 CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_blurred1, dev_kernel, radius, kernelLength);
	hipDeviceSynchronize();
	gaussianBlurPixel2 CUDA_KERNEL(blocks, threadsPerBlock)(dev_blurred1, dev_blurred2, dev_kernel, radius, kernelLength);
	hipDeviceSynchronize();

	return dev_blurred2;
}