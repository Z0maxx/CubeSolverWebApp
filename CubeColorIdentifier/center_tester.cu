#include "hip/hip_runtime.h"
#include "center_tester.cuh"

__constant__ const int const_testRange = 4;

__device__ __host__ bool checkSize(Center center)
{
	return center.size >= 250 && center.width > 5 && center.height > 5;
}

__global__ void testCenter(Center* dev_centers, int* length, int* dev_minY, int* dev_minX, int* dev_maxY, int* dev_maxX)
{
	int idx = threadIdx.x;
	Center center = dev_centers[idx];

	int i = 0;
	Center acceptedOtherCenter;
	Center acceptedTestedCenter;
	int acceptedhalfDiff = -1;

	if (!checkSize(center)) return;

	while (i < *length && !center.accepted)
	{
		Center otherCenter = dev_centers[i];
		if (i != idx &&
			checkSize(otherCenter) &&
			(
				center.y - const_testRange < otherCenter.y && center.y + const_testRange > otherCenter.y ||
				otherCenter.y - const_testRange < center.y && otherCenter.y + const_testRange > center.y
				))
		{
			int diff = abs(center.x - otherCenter.x);
			int halfDiff = diff / 2;
			int centerPoint = center.x < otherCenter.x ? center.x + halfDiff : otherCenter.x + halfDiff;
			int j = 0;
			while (j < *length && !center.accepted)
			{
				Center testedCenter = dev_centers[j];
				if (j != i && j != idx &&
					checkSize(testedCenter) &&
					centerPoint - const_testRange < testedCenter.x &&
					centerPoint + const_testRange > testedCenter.x &&
					(
						center.y - const_testRange < testedCenter.y && center.y + const_testRange > testedCenter.y ||
						otherCenter.y - const_testRange < testedCenter.y && otherCenter.y + const_testRange > testedCenter.y
						))
				{
					acceptedOtherCenter = otherCenter;
					acceptedTestedCenter = testedCenter;
					acceptedhalfDiff = halfDiff;

					center.accepted = true;
				}
				j++;
			}
		}
		i++;
	}

	if (center.accepted)
	{
		i = 0;
		center.accepted = false;
		while (i < *length && !center.accepted)
		{
			Center otherCenter = dev_centers[i];
			if (i != idx &&
				checkSize(otherCenter) &&
				(center.x - const_testRange < otherCenter.x && center.x + const_testRange > otherCenter.x ||
					otherCenter.x - const_testRange < center.x && otherCenter.x + const_testRange > center.x))
			{
				int diff = abs(center.y - otherCenter.y);
				int halfDiff = diff / 2;

				if (halfDiff - const_testRange < acceptedhalfDiff && halfDiff + const_testRange > acceptedhalfDiff ||
					acceptedhalfDiff - const_testRange < halfDiff && acceptedhalfDiff + const_testRange > halfDiff)
				{
					int centerPoint = center.y < otherCenter.y ? center.y + halfDiff : otherCenter.y + halfDiff;
					int j = 0;
					while (j < *length && !center.accepted)
					{
						Center testedCenter = dev_centers[j];
						if (j != i && j != idx &&
							checkSize(testedCenter) &&
							centerPoint - const_testRange < testedCenter.y && centerPoint + const_testRange > testedCenter.y &&
							(center.x - const_testRange < testedCenter.x && center.x + const_testRange > testedCenter.x ||
								otherCenter.x - const_testRange < testedCenter.x && otherCenter.x + const_testRange > testedCenter.x)
							)
						{
							atomicMin(dev_minY, center.y - center.height);
							atomicMin(dev_minY, otherCenter.y - otherCenter.height);

							atomicMin(dev_minX, center.x - center.width);
							atomicMin(dev_minX, otherCenter.x - otherCenter.width);
							atomicMin(dev_minX, testedCenter.x - testedCenter.width);

							atomicMax(dev_maxY, center.y + center.height);
							atomicMax(dev_maxY, otherCenter.y + otherCenter.height);

							atomicMax(dev_maxX, center.x + center.width);
							atomicMax(dev_maxX, otherCenter.x + otherCenter.width);
							atomicMax(dev_maxX, testedCenter.x + testedCenter.width);

							center.accepted = true;
						}
						j++;
					}
				}
			}
			i++;
		}
	}

	if (center.accepted)
	{
		atomicMin(dev_minY, acceptedOtherCenter.y - acceptedOtherCenter.height);
		atomicMin(dev_minY, acceptedTestedCenter.y - acceptedTestedCenter.height);

		atomicMin(dev_minX, acceptedOtherCenter.x - acceptedOtherCenter.width);

		atomicMax(dev_maxY, acceptedOtherCenter.y + acceptedOtherCenter.height);
		atomicMax(dev_maxY, acceptedTestedCenter.y + acceptedTestedCenter.height);

		atomicMax(dev_maxX, acceptedOtherCenter.x + acceptedOtherCenter.width);
	}
}

void testCenters(Center* dev_centers, int* length, int* minY, int* minX, int* maxY, int* maxX)
{
	*minY = height - 1;
	*minX = width - 1;
	*maxY = 0;
	*maxX = 0;
	int* dev_length;
	int* dev_minY;
	int* dev_minX;
	int* dev_maxY;
	int* dev_maxX;
	hipMalloc(&dev_length, sizeof(int*));
	hipMalloc(&dev_minY, sizeof(int*));
	hipMalloc(&dev_minX, sizeof(int*));
	hipMalloc(&dev_maxY, sizeof(int*));
	hipMalloc(&dev_maxX, sizeof(int*));
	hipMemcpy(dev_length, length, sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(dev_minX, minX, sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(dev_minY, minY, sizeof(int*), hipMemcpyHostToDevice);
	testCenter CUDA_KERNEL(1, *length)(dev_centers, dev_length, dev_minY, dev_minX, dev_maxY, dev_maxX);
	hipDeviceSynchronize();
	hipMemcpy(minX, dev_minX, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(minY, dev_minY, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(maxX, dev_maxX, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(maxY, dev_maxY, sizeof(int*), hipMemcpyDeviceToHost);
}