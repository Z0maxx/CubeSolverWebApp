#include "hip/hip_runtime.h"
#include "inside_image.cuh"

__global__ void setFromTop(int** dev_image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= const_width) return;

	dev_image[0][idx] = 50;
	int i = 0;
	while (i < const_height && dev_image[i][idx] != 255)
	{
		dev_image[i][idx] = 50;
		i++;
	}
}

__global__ void setFromBottom(int** dev_image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= const_width) return;

	dev_image[const_height - 1][idx] = 50;
	int i = const_height - 1;
	while (i > -1 && dev_image[i][idx] != 255)
	{
		dev_image[i][idx] = 50;
		i--;
	}
}

__global__ void setFromLeft(int** dev_image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= const_height) return;

	dev_image[idx][0] = 50;
	int i = 0;
	while (i < const_width && dev_image[idx][i] != 255)
	{
		dev_image[idx][i] = 50;
		i++;
	}
}

__global__ void setFromRight(int** dev_image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= const_height) return;

	dev_image[idx][const_width - 1] = 50;
	int i = const_width - 1;
	while (i > -1 && dev_image[idx][i] != 255)
	{
		dev_image[idx][i] = 50;
		i--;
	}
}

__global__ void setMissed(int** dev_image)
{
	int idx = getIdx();
	int y = getY(idx) + 1;
	int x = getX(idx) + 1;
	if (y < 1 || y >= const_height - 1 || x < 1 || x >= const_width - 1 || dev_image[y][x] != 50) return;
	
	if (dev_image[y][x + 1] == 0)
	{
		x++;
		while (dev_image[y][x] != 255)
		{
			dev_image[y][x] = 50;
			x++;
		}
	}
	x = getX(idx) + 1;
	if (dev_image[y][x - 1] == 0)
	{
		x--;
		while (dev_image[y][x] != 255)
		{
			dev_image[y][x] = 50;
			x--;
		}
	}
	x = getX(idx) + 1;
	if (dev_image[y + 1][x] == 0)
	{
		y++;
		while (dev_image[y][x] != 255)
		{
			dev_image[y][x] = 50;
			y++;
		}
	}
	y = getY(idx) + 1;
	if (dev_image[y - 1][x] == 0)
	{
		y--;
		while (dev_image[y][x] != 255)
		{
			dev_image[y][x] = 50;
			y--;
		}
	}
}

void insideImage(int** dev_image, int** image)
{
	int blocksTopBottom = ceil((float)width / threadsPerBlock);
	int blocksLeftRight = ceil((float)height / threadsPerBlock);
	setFromTop CUDA_KERNEL(blocksTopBottom, threadsPerBlock)(dev_image);
	setFromBottom CUDA_KERNEL(blocksTopBottom, threadsPerBlock)(dev_image);
	setFromLeft CUDA_KERNEL(blocksLeftRight, threadsPerBlock)(dev_image);
	setFromRight CUDA_KERNEL(blocksLeftRight, threadsPerBlock)(dev_image);
	hipDeviceSynchronize();
	int missedBlocks = ceil((float)(height - 2) * (width - 2) / threadsPerBlock);
	setMissed CUDA_KERNEL(missedBlocks, threadsPerBlock)(dev_image);
	hipDeviceSynchronize();
}