#include "hip/hip_runtime.h"
#include "mean_shift.cuh"

const int sample = 2;
__constant__ const int const_sample = 2;

__device__ int get3DDistance(int r1, int r2, int g1, int g2, int b1, int b2)
{
	return sqrt(pow(r1 - r2, 2) + pow(g1 - g2, 2) + pow(b1 - b2, 2));
}

__global__ void meanShiftPixel(int** dev_image, int** dev_temp, int bandwidth)
{
	int idx = getIdx();
	if (idx >= const_height / const_sample * const_width / const_sample) return;

	int ySample = idx / (const_width / const_sample);
	int y = ySample * const_sample;

	int xSample1 = idx % (const_width / const_sample);
	int xSample2 = xSample1 * 3;
	int x1 = xSample1 * const_sample;

	int x2 = x1 * 3;
	int tx = threadIdx.x;

	extern __shared__ MeanShiftVars shr_vars[];

	unsigned char blue1 = dev_image[y][x2 + 2];
	unsigned char green1 = dev_image[y][x2 + 1];
	unsigned char red1 = dev_image[y][x2];

	shr_vars[tx].meanShiftRed = 0;
	shr_vars[tx].meanShiftGreen = 0;
	shr_vars[tx].meanShiftBlue = 0;
	shr_vars[tx].count = 0;

	int bandwidthYM = y - bandwidth;
	int bandwidthYP = y + bandwidth;
	if (bandwidthYM < 0)
	{
		bandwidthYM = 0;
	}
	if (bandwidthYP >= const_height)
	{
		bandwidthYP = const_height - 1;
	}
	int bandwidthXM = x1 - bandwidth;
	int bandwidthXP = x1 + bandwidth;
	if (bandwidthXM < 0)
	{
		bandwidthXM = 0;
	}
	if (bandwidthXP >= const_width)
	{
		bandwidthXP = const_width - 1;
	}

	int count = 0;

	for (int i = bandwidthYM; i <= bandwidthYP; i++)
	{
		for (int j = bandwidthXM; j <= bandwidthXP; j++)
		{
			shr_vars[tx].rx = j * 3;
			shr_vars[tx].red2 = dev_image[i][shr_vars[tx].rx];
			shr_vars[tx].green2 = dev_image[i][shr_vars[tx].rx + 1];
			shr_vars[tx].blue2 = dev_image[i][shr_vars[tx].rx + 2];
			if (get3DDistance(red1, shr_vars[tx].red2, green1, shr_vars[tx].green2, blue1, shr_vars[tx].blue2) <= bandwidth)
			{
				shr_vars[tx].meanShiftRed += shr_vars[tx].red2 - red1;
				shr_vars[tx].meanShiftGreen += shr_vars[tx].green2 - green1;
				shr_vars[tx].meanShiftBlue += shr_vars[tx].blue2 - blue1;
				shr_vars[tx].count++;
			}
		}
	}

	dev_temp[ySample][xSample2] = (red1 + shr_vars[tx].meanShiftRed / shr_vars[tx].count);
	dev_temp[ySample][xSample2 + 1] = (green1 + shr_vars[tx].meanShiftGreen / shr_vars[tx].count);
	dev_temp[ySample][xSample2 + 2] = (blue1 + shr_vars[tx].meanShiftBlue / shr_vars[tx].count);
}

__global__ void setTempPixel(int** dev_image, int** dev_temp)
{
	int idx = getIdx();
	if (idx >= const_height / const_sample * const_width / const_sample) return;

	int ySample = idx / (const_width / const_sample);
	int y = ySample * const_sample;

	int xSample = idx % (const_width / const_sample) * 3;
	int x = xSample * const_sample;

	dev_image[y][x] = dev_temp[ySample][xSample];
	dev_image[y][x + 1] = dev_temp[ySample][xSample + 1];
	dev_image[y][x + 2] = dev_temp[ySample][xSample + 2];
}

__global__ void setPixelColor(int** dev_image, int** dev_meanShifted)
{
	int idx = getIdx();
	if (idx >= const_height / const_sample * const_width / const_sample) return;

	int y1 = idx / (const_width / const_sample);
	int y2 = y1 * const_sample;
	int x1 = idx % (const_width / const_sample);
	int x2 = x1 * 3;
	int x3 = x2 * const_sample;

	dev_meanShifted[y1][x2] = dev_image[y2][x3];
	dev_meanShifted[y1][x2 + 1] = dev_image[y2][x3 + 1];
	dev_meanShifted[y1][x2 + 2] = dev_image[y2][x3 + 2];
}

int** meanShift(int** dev_image)
{
	int meanShiftHeight = height / sample;
	int meanShiftHeightSize = meanShiftHeight * sizeof(int*);
	int meanShiftWidth = width / sample;
	int meanShiftWidthSize = meanShiftWidth * sizeof(int*) * 3;

	int meanShiftBlocks = ceil((float)height / sample * width / sample / threadsPerBlock);
	int** dev_temp;
	hipMalloc(&dev_temp, heightSize);
	int** temp = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&temp[i], widthRGBSize);
	}
	hipMemcpy(dev_temp, temp, heightSize, hipMemcpyHostToDevice);
	for (int i = 0; i < 3; i++)
	{
		meanShiftPixel CUDA_KERNEL(meanShiftBlocks, threadsPerBlock, threadsPerBlock * sizeof(MeanShiftVars))(dev_image, dev_temp, 20);
		error = hipDeviceSynchronize();
		setTempPixel  CUDA_KERNEL(meanShiftBlocks, threadsPerBlock)(dev_image, dev_temp);
		error = hipDeviceSynchronize();
	}
	
	int** dev_meanShifted;
	hipMalloc(&dev_meanShifted, meanShiftHeightSize);
	int** meanShifted = (int**)malloc(meanShiftHeightSize);
	for (int i = 0; i < meanShiftHeight; i++)
	{
		hipMalloc(&meanShifted[i], meanShiftWidthSize);
	}
	hipMemcpy(dev_meanShifted, meanShifted, meanShiftHeightSize, hipMemcpyHostToDevice);

	setPixelColor CUDA_KERNEL(meanShiftBlocks, threadsPerBlock)(dev_image, dev_meanShifted);
	hipDeviceSynchronize();

	return dev_meanShifted;
}