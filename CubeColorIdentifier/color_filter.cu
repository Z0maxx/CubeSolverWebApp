#include "hip/hip_runtime.h"
#include "color_filter.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__global__ void filterColorPixel(int** dev_image, int** dev_segmented, int** dev_filtered)
{
	int idx = getIdx();
	if (idx >= const_imageLength) return;
	int y = getY(idx);
	int x1 = getX(idx);
	int x2 = x1 * 3;
	float blue = dev_image[y][x2 + 2];
	float green = dev_image[y][x2 + 1];
	float red = dev_image[y][x2];
	CubeColor cubeColor = convertRgbToCubeColor(red, green, blue);
	if (cubeColor == CubeColor_None || dev_segmented[y][x1] == 255)
	{
		dev_filtered[y][x1] = 255;
	}
}

int** filterColor(int** dev_image, int** dev_segmented, int** filtered)
{
	int** dev_filtered;
	hipMalloc(&dev_filtered, heightSize);
	
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&filtered[i], widthIntSize);
	}
	hipMemcpy(dev_filtered, filtered, heightSize, hipMemcpyHostToDevice);

	filterColorPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_segmented, dev_filtered);
	hipDeviceSynchronize();

	return dev_filtered;
}