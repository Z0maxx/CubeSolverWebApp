#include "hip/hip_runtime.h"
#include "reduce.cuh"

__global__ void reducePixel(int** dev_thresholded, int** dev_reducedThresholded, float** dev_angles, float** dev_reducedAngles, int** dev_baseImage, Color** dev_reducedBaseImage, int minX, int minY)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	dev_reducedThresholded[y][x] = dev_thresholded[y + minY][x + minX];
	dev_reducedAngles[y][x] = dev_angles[y + minY][x + minX];
	dev_reducedBaseImage[y][x] = { dev_baseImage[y + minY][(x + minX) * 3], dev_baseImage[y + minY][(x + minX) * 3 + 1], dev_baseImage[y + minY][(x + minX) * 3 + 2] };
}

void reduce(int** dev_thresholded, int** dev_reducedThresholded, float** dev_angles, float** dev_reducedAngles, int** dev_baseImage, Color** reducedBaseImage, Color** dev_reducedBaseImage, int minX, int minY)
{
	int** reducedThresholded = (int**)malloc(heightSize);
	float** reducedAngles = (float**)malloc(heightSize);
	int widthColorSize = width * sizeof(Color);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&reducedThresholded[i], widthIntSize);
		hipMalloc(&reducedAngles[i], widthDoubleSize);
		hipMalloc(&reducedBaseImage[i], widthColorSize);
	}

	hipMemcpy(dev_reducedThresholded, reducedThresholded, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_reducedAngles, reducedAngles, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_reducedBaseImage, reducedBaseImage, heightSize, hipMemcpyHostToDevice);

	reducePixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_thresholded, dev_reducedThresholded, dev_angles, dev_reducedAngles, dev_baseImage, dev_reducedBaseImage, minX, minY);
	hipDeviceSynchronize();
}