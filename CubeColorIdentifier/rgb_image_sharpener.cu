#include "hip/hip_runtime.h"
#include "rgb_image_sharpener.cuh"

__device__ int averageFilterPixel(int** dev_image, int y, int x1, int x2, int offset)
{
	int sum = dev_image[y][x2 + offset];

	if (x1 > 0)
	{
		sum += dev_image[y][x2 - 3 + offset];
	}

	if (x1 < const_width - 1)
	{
		sum += dev_image[y][x2 + 3 + offset];
	}

	if (y > 0)
	{
		sum += dev_image[y - 1][x2 + offset];
		if (x1 > 0)
		{
			sum += dev_image[y - 1][x2 - 3 + offset];
		}

		if (x1 < const_width - 1)
		{
			sum += dev_image[y - 1][x2 + 3 + offset];
		}
	}
	if (y < const_height - 1)
	{
		sum += dev_image[y + 1][x2 + offset];
		if (x1 > 0)
		{
			sum += dev_image[y + 1][x2 - 3 + offset];
		}
		if (x1 < const_width - 1)
		{
			sum += dev_image[y + 1][x2 + 3 + offset];
		}
	}

	return sum / 9;
}

__global__ void smoothPixel(int** dev_image, int** dev_smoothed)
{
	int idx = getIdx();
	if (idx >= const_imageLength) return;

	int y = getY(idx);
	int x1 = getX(idx);
	int x2 = x1 * 3;

	for (int i = 0; i < 3; i++)
	{
		int color = averageFilterPixel(dev_image, y, x1, x2, i);
		if (color < 0)
		{
			color = 0;
		}
		else if (color > 255)
		{
			color = 255;
		}
		dev_smoothed[y][x2 + i] = color;
	}
}

__device__ int laplacePixel(int** dev_image, int y, int x1, int x2, int offset)
{
	int sum = dev_image[y][x2 + offset] * (-9);

	if (x1 > 0)
	{
		sum += dev_image[y][x2 - 3 + offset];
	}

	if (x1 < const_width - 1)
	{
		sum += dev_image[y][x2 + 3 + offset];
	}

	if (y > 0)
	{
		sum += dev_image[y - 1][x2 + offset];
		if (x1 > 0)
		{
			sum += dev_image[y - 1][x2 - 3 + offset];
		}

		if (x1 < const_width - 1)
		{
			sum += dev_image[y - 1][x2 + 3 + offset];
		}
	}
	if (y < const_height - 1)
	{
		sum += dev_image[y + 1][x2 + offset];
		if (x1 > 0)
		{
			sum += dev_image[y + 1][x2 - 3 + offset];
		}
		if (x1 < const_width - 1)
		{
			sum += dev_image[y + 1][x2 + 3 + offset];
		}
	}

	return -sum;
}

__global__ void sharpenPixel(int** dev_image, int** dev_sharpened)
{
	int idx = getIdx();
	if (idx >= const_imageLength) return;

	int y = getY(idx);
	int x1 = getX(idx);
	int x2 = x1 * 3;

	for (int i = 0; i < 3; i++)
	{
		int color = laplacePixel(dev_image, y, x1, x2, i);
		if (color < 0)
		{
			color = 0;
		}
		else if (color > 255)
		{
			color = 255;
		}
		dev_sharpened[y][x2 + i] = color;
	}
}

int** sharpenImage(int** dev_image)
{
	int** dev_smoothed;
	int** dev_sharpened;
	hipMalloc(&dev_smoothed, heightSize);
	hipMalloc(&dev_sharpened, heightSize);
	int** sharpened = (int**)malloc(heightSize);
	int** smoothed = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&smoothed[i], widthRGBSize);
		hipMalloc(&sharpened[i], widthRGBSize);
	}
	hipMemcpy(dev_smoothed, smoothed, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_sharpened, sharpened, heightSize, hipMemcpyHostToDevice);

	smoothPixel	CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_smoothed);
	hipDeviceSynchronize();

	sharpenPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_smoothed, dev_sharpened);
	hipDeviceSynchronize();

	return dev_sharpened;
}