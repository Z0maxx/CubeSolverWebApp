#include "hip/hip_runtime.h"
#include "cube_centers.cuh"

__global__ void squareCenterPixel(int** dev_image, Center* dev_centers, int* dev_maxSquareWidth, int* dev_maxSquareHeight, unsigned int* dev_count)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength || dev_image[y][x] != 0) return;

	int left = 0;
	int right = 0;
	int top = 0;
	int bottom = 0;
	while (x - left - 1 >= 0 && dev_image[y][x - left - 1] != 255)
	{
		left++;
	}
	while (x + right + 1 < const_width && dev_image[y][x + right + 1] != 255)
	{
		right++;
	}
	while (y - top - 1 >= 0 && dev_image[y - top - 1][x] != 255)
	{
		top++;
	}
	while (y + bottom + 1 < const_height && dev_image[y + bottom + 1][x] != 255)
	{
		bottom++;
	}
	if (left > 1 && right > 1 && top > 1 && bottom > 1 && abs(left - right) <= 1 && abs(top - bottom) <= 1)
	{
		int halfHeight = top * 0.7;
		int halfWidth = left * 0.7;
		int height = 2 * halfHeight + 1;
		int width = 2 * halfWidth + 1;
		atomicMax(dev_maxSquareHeight, height);
		atomicMax(dev_maxSquareWidth, width);
		dev_image[y][x] = 100;
		int centerIdx = atomicAdd(dev_count, 1);
		dev_centers[centerIdx].y = y;
		dev_centers[centerIdx].x = x;
		dev_centers[centerIdx].halfHeight = halfHeight;
		dev_centers[centerIdx].halfWidth = halfWidth;
		dev_centers[centerIdx].height = height;
		dev_centers[centerIdx].width = width;
		dev_centers[centerIdx].size = width * height;
	}
}

__global__ void squareSingleCenter(int** dev_image, Center* dev_centers, Center* dev_filteredCenters, unsigned int* dev_filteredCount)
{
	Center center = dev_centers[threadIdx.x];
	int x = center.x;
	int y = center.y;

	if (dev_image[y][x - 1] != 100 && 
		dev_image[y - 1][x] != 100 && 
		dev_image[y - 1][x - 1] != 100 &&
		dev_image[y + 1][x - 1] != 100)
	{
		dev_filteredCenters[atomicAdd(dev_filteredCount, 1)] = center;
		for (int i = center.y - 5; i <= center.y + 5; i++)
		{
			if (i >= 0 && i < const_height)
			{
				dev_image[i][x] = 125;
			}
		}
		for (int i = center.x - 5; i <= center.x + 5; i++)
		{
			if (i >= 0 && i < const_width)
			{
				dev_image[y][i] = 125;
			}
		}
	}
	else
	{
		dev_image[y][x] = 0;
	}
}

int compCenter1(const void* a, const void* b)
{
	return ((dim3*)a)->y - ((dim3*)b)->y;
}

bool compCenter2(dim3 center1, dim3 center2)
{
	return center1.x < center2.x;
}

Center* cubeCenters(int** dev_image, int* maxSquareHeight, int* maxSquareWidth, int* length)
{
	int* dev_maxSquareHeight;
	int* dev_maxSquareWidth;
	hipMalloc(&dev_maxSquareHeight, sizeof(int*));
	hipMalloc(&dev_maxSquareWidth, sizeof(int*));
	Center* dev_centers;
	unsigned int* dev_count;
	hipMalloc(&dev_count, sizeof(unsigned int*));
	int centersSize = width * height * sizeof(Center);
	hipMalloc(&dev_centers, centersSize);
	squareCenterPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_centers, dev_maxSquareHeight, dev_maxSquareWidth, dev_count);
	hipDeviceSynchronize();
	hipMemcpy(maxSquareHeight, dev_maxSquareHeight, sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(maxSquareWidth, dev_maxSquareWidth, sizeof(int*), hipMemcpyDeviceToHost);
	int* count = new int;
	hipMemcpy(count, dev_count, sizeof(int*), hipMemcpyDeviceToHost);

	centersSize = *count * sizeof(Center);
	Center* dev_filteredCenters;
	hipMalloc(&dev_filteredCenters, centersSize);
	unsigned int* dev_filteredCount;
	hipMalloc(&dev_filteredCount, sizeof(unsigned int*));
	squareSingleCenter CUDA_KERNEL(1, *count)(dev_image, dev_centers, dev_filteredCenters, dev_filteredCount);
	hipDeviceSynchronize();
	hipMemcpy(length, dev_filteredCount, sizeof(int*), hipMemcpyDeviceToHost);

	return dev_filteredCenters;
}