#include "hip/hip_runtime.h"
#include "sobel_filter.cuh"

__constant__ int const_kernelAvg5x5[5] = { 1, 4, 6, 4, 1 };
__constant__ int const_kernelDiff5x5[5] = { -1, -2, 0, 2, 1 };

__global__ void sobelXAvg5x5Pixel(int** dev_image, int** dev_sobelXAvg)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	int sum = 0;
	int i = y - 2;
	for (int j = 0; j < 5; j++)
	{
		if (i >= 0 && i < const_height)
		{
			sum += dev_image[i][x] * const_kernelAvg5x5[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_image[-i][x];
			}
			else
			{
				mirror = dev_image[const_height - (i - const_height + 2)][x];
			}
			sum += mirror * const_kernelAvg5x5[j];
		}
		i++;
	}
	dev_sobelXAvg[y][x] = sum;
}

__global__ void sobelYAvg5x5Pixel(int** dev_image, int** dev_sobelYAvg)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	int sum = 0;
	int i = x - 2;
	for (int j = 0; j < 5; j++)
	{
		if (i >= 0 && i < const_width)
		{
			sum += dev_image[y][i] * const_kernelAvg5x5[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_image[y][-i];
			}
			else
			{
				mirror = dev_image[y][const_width - (i - const_width + 2)];
			}
			sum += mirror * const_kernelAvg5x5[j];
		}
		i++;
	}
	dev_sobelYAvg[y][x] = sum;
}

__global__ void sobelXDiff5x5Pixel(int** dev_sobelXAvg, int** dev_sobelXDiff)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	int sum = 0;
	int i = x - 2;
	for (int j = 0; j < 5; j++)
	{
		if (i >= 0 && i < const_width)
		{
			sum += dev_sobelXAvg[y][i] * const_kernelDiff5x5[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_sobelXAvg[y][-i];
			}
			else
			{
				mirror = dev_sobelXAvg[y][const_width - (i - const_width + 2)];
			}
			sum += mirror * const_kernelDiff5x5[j];
		}
		i++;
	}
	dev_sobelXDiff[y][x] = sum;
}

__global__ void sobelYDiff5x5Pixel(int** dev_sobelYAvg, int** dev_sobelYDiff)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	int sum = 0;
	int i = y - 2;
	for (int j = 0; j < 5; j++)
	{
		if (i >= 0 && i < const_height)
		{
			sum += dev_sobelYAvg[i][x] * const_kernelDiff5x5[j];
		}
		else
		{
			int mirror = 0;
			if (i < 0)
			{
				mirror = dev_sobelYAvg[-i][x];
			}
			else
			{
				mirror = dev_sobelYAvg[const_height - (i - const_height + 2)][x];
			}
			sum += mirror * const_kernelDiff5x5[j];
		}
		i++;
	}
	dev_sobelYDiff[y][x] = sum;
}

__global__ void sobelGradientPixel(int** dev_sobelXDiff, int** dev_sobelYDiff, int** dev_sobelGradient)
{
	int idx = getIdx();
	int y = getY(idx);
	int x = getX(idx);
	if (idx >= const_imageLength) return;

	dev_sobelGradient[y][x] = sqrt(pow((float)dev_sobelXDiff[y][x], 2) + pow((float)dev_sobelYDiff[y][x], 2));
}

int** sobelX5x5(int** dev_image)
{
	int** dev_sobelXAvg;
	int** dev_sobelXDiff;
	hipMalloc(&dev_sobelXAvg, heightSize);
	hipMalloc(&dev_sobelXDiff, heightSize);
	int** sobelXAvg = (int**)malloc(heightSize);
	int** sobelXDiff = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&sobelXAvg[i], widthIntSize);
		hipMalloc(&sobelXDiff[i], widthIntSize);
	}
	hipMemcpy(dev_sobelXAvg, sobelXAvg, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_sobelXDiff, sobelXDiff, heightSize, hipMemcpyHostToDevice);

	sobelXAvg5x5Pixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_image, dev_sobelXAvg);
	hipDeviceSynchronize();
	sobelXDiff5x5Pixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_sobelXAvg, dev_sobelXDiff);
	hipDeviceSynchronize();

	return dev_sobelXDiff;
}

int** sobelY5x5(int** dev_image)
{
	int** dev_sobelYAvg;
	int** dev_sobelYDiff;
	hipMalloc(&dev_sobelYAvg, heightSize);
	hipMalloc(&dev_sobelYDiff, heightSize);
	int** sobelYAvg = (int**)malloc(heightSize);
	int** sobelYDiff = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&sobelYAvg[i], widthIntSize);
		hipMalloc(&sobelYDiff[i], widthIntSize);
	}
	hipMemcpy(dev_sobelYAvg, sobelYAvg, heightSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_sobelYDiff, sobelYDiff, heightSize, hipMemcpyHostToDevice);


	sobelYAvg5x5Pixel CUDA_KERNEL(blocks, threadsPerBlock) (dev_image, dev_sobelYAvg);
	hipDeviceSynchronize();
	sobelYDiff5x5Pixel CUDA_KERNEL(blocks, threadsPerBlock) (dev_sobelYAvg, dev_sobelYDiff);
	hipDeviceSynchronize();

	return dev_sobelYDiff;
}

int** sobel(int** dev_sobelX, int** dev_sobelY)
{
	int** dev_sobelGradient;
	hipMalloc(&dev_sobelGradient, heightSize);
	int** sobelGradient = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		hipMalloc(&sobelGradient[i], widthIntSize);
	}
	hipMemcpy(dev_sobelGradient, sobelGradient, heightSize, hipMemcpyHostToDevice);

	sobelGradientPixel CUDA_KERNEL(blocks, threadsPerBlock)(dev_sobelX, dev_sobelY, dev_sobelGradient);
	hipDeviceSynchronize();

	return dev_sobelGradient;
}