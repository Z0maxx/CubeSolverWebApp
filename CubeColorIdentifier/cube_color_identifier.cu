#include "hip/hip_runtime.h"
#include "cube_color_identifier.cuh"

__global__ void setBasePixel(int** dev_baseImage, int* dev_image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= const_imageLength * 3) return;
	int y = idx / (const_width * 3);
	int x = idx % (const_width * 3);
	dev_baseImage[y][x] = dev_image[idx];
}

void setVariables(int newWidth, int newHeight)
{
	height = newHeight;
	width = newWidth;
	heightSize = height * sizeof(int*);
	widthIntSize = width * sizeof(int);
	widthDoubleSize = width * sizeof(float);
	widthRGBSize = widthIntSize * 3;
	imageLength = height * width;
	blocks = ceil((float)imageLength / threadsPerBlock);
	baseImageLength = imageLength * 3;

	hipMemcpyToSymbol(HIP_SYMBOL(const_width), &width, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_height), &height, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(const_imageLength), &imageLength, sizeof(int));
}

void identifyColors(int* image)
{
	int** dev_baseImage;
	hipMalloc(&dev_baseImage, heightSize);
	int* dev_image;
	int size = height * width * 3 * sizeof(int);
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, image, size, hipMemcpyHostToDevice);
	int** baseImage = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		if (baseImage != nullptr)
		{
			hipMalloc(&baseImage[i], widthRGBSize);
		}
	}
	hipMemcpy(dev_baseImage, baseImage, heightSize, hipMemcpyHostToDevice);
	int blocksBaseImage = ceil((float)width * height * 3 / threadsPerBlock);
	setBasePixel CUDA_KERNEL(blocksBaseImage, threadsPerBlock)(dev_baseImage, dev_image);
	hipDeviceSynchronize();

	int** dev_sharpened = sharpenImage(dev_baseImage);
	int** dev_meanShifted = meanShift(dev_sharpened);

	setVariables(width / sample, height / sample);

	int** dev_grayscaled = grayscale(dev_meanShifted);
	int** dev_blurred1 = gaussianBlur(dev_grayscaled, 1.2);
	int** dev_blurred2 = gaussianBlur(dev_grayscaled, 1.5);
	int** dev_segmented = segmentImage(dev_blurred1);
	int** filtered = (int**)malloc(heightSize);
	int** dev_filtered = filterColor(dev_meanShifted, dev_segmented, filtered);
	int* maxSquareHeight = new int;
	int* maxSquareWidth = new int;
	int* length = new int;
	Center* dev_centers = cubeCenters(dev_filtered, maxSquareHeight, maxSquareWidth, length);

	int* minX = new int;
	int* minY = new int;
	int* maxX = new int;
	int* maxY = new int;
	testCenters(dev_centers, length, minY, minX, maxY, maxX);
	if (*maxX > *minX && *maxY > *minY)
	{
		*minX -= 10;
		*minY -= 10;
		*maxX += 10;
		*maxY += 10;
		if (*minX < 0) *minX = 0;
		if (*minY < 0) *minY = 0;
		if (*maxX >= width) *maxX = width - 1;
		if (*maxY >= height) *maxY = height - 1;
		setVariables(*maxX - *minX + 1, *maxY - *minY + 1);
		int** dev_reducedBlurred;
		hipMalloc(&dev_reducedBlurred, heightSize);
		int** dev_reducedMeanShifted;
		hipMalloc(&dev_reducedMeanShifted, heightSize);
		reduceMeanShifted(dev_blurred2, dev_reducedBlurred, dev_meanShifted, dev_reducedMeanShifted, *minX, *minY);
		int** dev_sobelX = sobelX5x5(dev_reducedBlurred);
		int** dev_sobelY = sobelY5x5(dev_reducedBlurred);
		int** dev_sobel = sobel(dev_sobelX, dev_sobelY);
		float** dev_angles = gradientAngle(dev_sobelX, dev_sobelY);
		int** dev_supressed = nonMaximumSupression(dev_sobelX, dev_sobelY, dev_sobel, dev_angles);
		int** thresholded = (int**)malloc(heightSize);
		int** dev_thresholded = hysteresisThreshold(dev_supressed, thresholded, 100, 550);
		std::vector<HoughLine> houghLines = houghTransform(dev_thresholded, dev_angles, 30);
		int** resHoughImage = (int**)malloc(heightSize);
		int** dev_houghImage = houghImage(houghLines, resHoughImage, thresholded, dev_thresholded);
		insideImage(dev_houghImage, resHoughImage);
		minMaxCoords(dev_houghImage, minX, minY, maxX, maxY);
		setVariables(*maxX - *minX + 1, *maxY - *minY + 1);

		if (width > 0 && height > 0)
		{
			printf("%d %d\n", width, height);
			int** dev_reducedThresholded;
			float** dev_reducedAngles;
			hipMalloc(&dev_reducedThresholded, heightSize);
			hipMalloc(&dev_reducedAngles, heightSize);
			Color** dev_reducedBaseImage;
			hipMalloc(&dev_reducedBaseImage, heightSize);
			Color** reducedBaseImage = (Color**)malloc(heightSize);
			int** dev_reducedSegmentedImage;
			hipMalloc(&dev_reducedSegmentedImage, heightSize);
			reduce(dev_thresholded, dev_reducedThresholded, dev_angles, dev_reducedAngles, dev_reducedMeanShifted, reducedBaseImage, dev_reducedBaseImage, *minX, *minY);
			houghLines = houghTransform(dev_reducedThresholded, dev_reducedAngles, 30);
			int** reducedHoughImage = (int**)malloc(heightSize);
			int** dev_reducedHoughImage = houghImage(houghLines, reducedHoughImage, thresholded, dev_reducedThresholded);
			insideImage(dev_reducedHoughImage, reducedHoughImage);
			dev_centers = cubeCenters(dev_reducedHoughImage, maxSquareHeight, maxSquareWidth, length);

			std::vector<Center> testedCenters = testColors(dev_centers, reducedBaseImage, dev_reducedBaseImage, *maxSquareWidth, *maxSquareHeight, *length);
			if (testedCenters.size() >= 9)
			{
				std::vector<std::string> colors = cubeColors(testedCenters);
				std::string colorsStr = "";
				for (int i = 0; i < 8; i++)
				{
					colorsStr += colors[i] + "|";
				}
				colorsStr += colors[8];
				printf("%s", colorsStr.c_str());
			}
			else
			{
				printf("||||||||");
			}
		}
		else
		{
			printf("%d %d\n", 0, 0);
			printf("||||||||");
		}
	}
	else
	{
		printf("%d %d\n", 0, 0);
		printf("||||||||");
	}
}