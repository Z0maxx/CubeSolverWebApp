#include "variables.cuh"

__constant__ int const_width;
__constant__ int const_height;
__constant__ int const_imageLength;

__device__ int getIdx()
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int getY(int idx)
{
	return idx / const_width;
}

__device__ int getX(int idx)
{
	return idx % const_width;
}

int width;
int height;
int imageLength;
int baseImageLength;
int heightSize;
int widthIntSize;
int widthDoubleSize;
int widthRGBSize;
int threadsPerBlock;
int blocks;
hipError_t error;
char* fileName;