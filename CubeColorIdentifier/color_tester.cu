#include "hip/hip_runtime.h"
#include "color_tester.cuh"

__device__ Color translateToRGB(CubeColor color)
{
	switch (color)
	{
	case (CubeColor_Red):
		return { 255, 0, 0 };
	case (CubeColor_Green):
		return { 0, 255, 0 };
	case (CubeColor_Orange):
		return { 255, 125, 0 };
	case (CubeColor_Yellow):
		return { 255, 255, 0 };
	case (CubeColor_Blue):
		return { 0, 0, 255 };
	case (CubeColor_White):
		return { 255, 255, 255 };
	case (CubeColor_None):
		return { 0,0,0 };
	}
}

__global__ void setPixelColor(Center* dev_centers, Color** dev_image, unsigned int** dev_cubeColorCounts)
{
	Center center = dev_centers[blockIdx.x];
	int idx = (blockIdx.y * blockDim.x + threadIdx.x);
	if (idx >= center.size) return;

	int y = idx / center.width;
	int x = idx % center.width;

	int displacedY = center.y - (center.halfHeight - y);
	int displacedX = center.x - (center.halfWidth - x);
	Color color = dev_image[displacedY][displacedX];
	CubeColor cubeColor = convertRgbToCubeColor(color.r, color.g, color.b);
	Color rgb = translateToRGB(cubeColor);
	dev_image[displacedY][displacedX].r = rgb.r;
	dev_image[displacedY][displacedX].g = rgb.g;
	dev_image[displacedY][displacedX].b = rgb.b;
	atomicAdd(&dev_cubeColorCounts[blockIdx.x][cubeColor], 1);
}

void testCenter(Center* centers, unsigned int** cubeColorCounts, std::vector<Center>* centerVec, int threadId)
{
	Center center = centers[threadId];
	unsigned int* counts = cubeColorCounts[threadId];
	if (counts[0] <= 0.05 * center.size)
	{
		int differentColors = 0;
		int max = 0;
		for (int i = 1; i < 7; i++)
		{
			if (counts[i] > 0)
			{
				differentColors++;
			}
			if (counts[i] > counts[max])
			{
				max = i;
			}
		}
		if (differentColors <= 2)
		{
			center.accepted = true;
			center.mainColor = (CubeColor)max;
		}
	}
	
	(*centerVec)[threadId] = center;
}

bool compCenter(Center center1, Center center2)
{
	return center1.y < center2.y;
}

std::vector<Center> testColors(Center* dev_centers, Color** image, Color** dev_image, int maxWidth, int maxHeight, int length)
{
	int widthHeight = maxWidth * maxHeight;
	int lengthSize = length * sizeof(unsigned int**);
	unsigned int** dev_cubeColorCounts;
	hipMalloc(&dev_cubeColorCounts, lengthSize);
	CubeColor** cubeColorCounts = (CubeColor**)malloc(lengthSize);
	int size = 7 * sizeof(unsigned int*);
	for (int i = 0; i < length; i++)
	{
		hipMalloc(&cubeColorCounts[i], size);
	}
	hipMemcpy(dev_cubeColorCounts, cubeColorCounts, lengthSize, hipMemcpyHostToDevice);

	int centerBlocks = ceil((float)widthHeight / threadsPerBlock);
	setPixelColor CUDA_KERNEL(dim3(length, centerBlocks), widthHeight > threadsPerBlock ? threadsPerBlock : widthHeight)(dev_centers, dev_image, dev_cubeColorCounts);
	hipDeviceSynchronize();

	unsigned int** resCubeColorCounts = (unsigned int**)malloc(lengthSize);
	for (int i = 0; i < length; i++)
	{
		resCubeColorCounts[i] = (unsigned int*)malloc(size);
		hipMemcpy(resCubeColorCounts[i], cubeColorCounts[i], size, hipMemcpyDeviceToHost);
	}

	int centerSize = length * sizeof(Center);
	Center* centers = (Center*)malloc(centerSize);
	hipMemcpy(centers, dev_centers, centerSize, hipMemcpyDeviceToHost);

	std::vector<std::thread> threads;
	std::vector<Center> centerVec(length);
	for (int i = 0; i < length; i++)
	{
		int id = i;
		threads.emplace_back(std::thread(testCenter, centers, resCubeColorCounts, &centerVec, id));
	}

	for (auto& thread : threads)
	{
		thread.join();
	}

	std::sort(centerVec.begin(), centerVec.end(), compCenter);

	int** res = (int**)malloc(heightSize);
	for (int i = 0; i < height; i++)
	{
		res[i] = (int*)malloc(widthRGBSize);
		hipMemcpy(res[i], image[i], widthRGBSize, hipMemcpyDeviceToHost);
	}

	FILE* f = fopen(fileName, "wb");
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			fprintf(f, "%c%c%c", res[i][j * 3 + 2], res[i][j * 3 + 1], res[i][j * 3]);
		}
	}
	fclose(f);

	return centerVec;
}