#include "ms_clock.cuh"

hipEvent_t start, stop;

void startTimer()
{
	hipEventCreate(&start);
	hipEventRecord(start, 0);
}

void stopTimer()
{
	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
}

float getElapsedTime()
{
	float time;
	hipEventElapsedTime(&time, start, stop);
	return time;
}
